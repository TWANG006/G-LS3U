#include "hip/hip_runtime.h"
#include "aia_cudaf.h"
#include <time.h>
#include <functional>
#include <random>
#include <omp.h>

#include ""

namespace AIA{

/*---------------------------------------CUDA Kernels----------------------------------*/
__global__
void generate_csrValA1_rhs1_kernel(float* d_out_csrValA1,
								   float* d_out_csr_rhs1,
								   uchar* d_in_img,
								   float* d_in_delta,
								   const int M,
								   const int N)
{
	for (int j = threadIdx.x + blockDim.x *blockIdx.x;
		 j < N;
		 j += blockDim.x * gridDim.x)
	{
		float a3 = 0, a4 = 0, a6 = 0, a7 = 0, a8 = 0;
		float b0 = 0, b1 = 0, b2 = 0;

		for (int i = 0; i < M; i++)
		{
			float delta = d_in_delta[i];
			float cos_delta = cos(delta);
			float sin_delta = sin(delta);
			float Iij = static_cast<float>(d_in_img[i*N + j]);

			a3 += cos_delta;
			a4 += cos_delta*cos_delta;
			a6 += sin_delta;
			a7 += sin_delta*cos_delta;
			a8 += sin_delta*sin_delta;
			b0 += Iij;
			b1 += Iij * cos_delta;
			b2 += Iij * sin_delta;
		}

		d_out_csrValA1[j * 9 + 0] = M;
		d_out_csrValA1[j * 9 + 1] = a3;
		d_out_csrValA1[j * 9 + 2] = a6;
		d_out_csrValA1[j * 9 + 3] = a3;
		d_out_csrValA1[j * 9 + 4] = a4;
		d_out_csrValA1[j * 9 + 5] = a7;
		d_out_csrValA1[j * 9 + 6] = a6;
		d_out_csrValA1[j * 9 + 7] = a7;
		d_out_csrValA1[j * 9 + 8] = a8;

		d_out_csr_rhs1[j * 3 + 0] = b0;
		d_out_csr_rhs1[j * 3 + 1] = b1;
		d_out_csr_rhs1[j * 3 + 2] = b2;
	}
}

__global__
void generate_csrColIndA1_csrRowPtrA1_kernel(int* d_out_csrColIndA1,
											 int* d_out_csrRowPtrA1,
											 const int N)
{
	for (int i = threadIdx.x + blockIdx.x * blockDim.x;
		 i < N;
		 i += blockDim.x * gridDim.x)
	{
		int a1 = i * 3;
		
		d_out_csrRowPtrA1[i * 3 + 0] = a1;
		d_out_csrRowPtrA1[i * 3 + 1] = a1 + 3;
		d_out_csrRowPtrA1[i * 3 + 2] = a1 + 6;

		int a2 = a1 + 1;
		int a3 = a1 + 2;

		d_out_csrColIndA1[i * 9 + 0] = a1;
		d_out_csrColIndA1[i * 9 + 1] = a2;
		d_out_csrColIndA1[i * 9 + 2] = a3;
		d_out_csrColIndA1[i * 9 + 3] = a1;
		d_out_csrColIndA1[i * 9 + 4] = a2;
		d_out_csrColIndA1[i * 9 + 5] = a3;
		d_out_csrColIndA1[i * 9 + 6] = a1;
		d_out_csrColIndA1[i * 9 + 7] = a2;
		d_out_csrColIndA1[i * 9 + 8] = a3;
	}

	// Last ele of csrRowIndA is nnz + csrRowIndA(0)
	if (blockIdx.x == 0 && threadIdx.x ==0)
		d_out_csrRowPtrA1[3 * N] = 9 * N;
}

/*--------------------------------------End CUDA Kernels--------------------------------*/

AIA_CUDAF::AIA_CUDAF(const std::vector<cv::Mat>& v_f)
	: m_d_img(nullptr)
	, m_d_csrColIndA1(nullptr)
	, m_d_csrValA1(nullptr)
	, m_d_csrRowPtrA1(nullptr)
	, m_d_b1(nullptr)
	, m_d_phi(nullptr)
	, m_d_delta(nullptr)
	, m_h_delta(nullptr)
	, m_h_A2(nullptr)
	, m_h_b2(nullptr)
{
	// Get params
	m_M = v_f.size();
	m_N = v_f[0].cols*v_f[0].rows;
	m_cols = v_f[0].cols;
	m_rows = v_f[0].rows;

	// Allocate required pinned host memory
	m_h_old_delta = (float*)malloc(sizeof(float)*m_M);
	WFT_FPA::Utils::cucreateptr(m_h_delta, sizeof(float)*m_M);
	WFT_FPA::Utils::cucreateptr(m_h_A2, sizeof(float) * 9);
	WFT_FPA::Utils::cucreateptr(m_h_b2, sizeof(float)*m_M * 3);

	// Allocate device memory
	checkCudaErrors(hipMalloc((void**)&m_d_csrValA1, sizeof(float) * 9 * m_N));
	checkCudaErrors(hipMalloc((void**)&m_d_csrRowPtrA1, sizeof(int)*(3 * m_N + 1)));
	checkCudaErrors(hipMalloc((void**)&m_d_csrColIndA1, sizeof(int) * 9 * m_N));
	checkCudaErrors(hipMalloc((void**)&m_d_b1, sizeof(float) * 3 * m_N));
	checkCudaErrors(hipMalloc((void**)&m_d_phi, sizeof(float) * m_cols * m_rows));
	checkCudaErrors(hipMalloc((void**)&m_d_delta, sizeof(float) * m_M));
	checkCudaErrors(hipMalloc((void**)&m_d_img, sizeof(uchar)*m_M*m_N));

	// Initialize the csrRowPtrA & csrColIndA here because they remain at the same patterns
	generate_csrColIndA1_csrRowPtrA1_kernel<<<8*32, 256>>>(m_d_csrColIndA1, m_d_csrRowPtrA1, m_N);
	getLastCudaError("generate_csrColIndA1_csrRowPtrA1_kernel launch failed!");

	// Create cuSolver required handles
	checkCudaErrors(hipsolverSpCreate(&m_cuSolverHandle));
	checkCudaErrors(hipStreamCreate(&m_stream));
	checkCudaErrors(hipsolverSpSetStream(m_cuSolverHandle, m_stream));
	checkCudaErrors(hipsparseCreateMatDescr(&m_desrA));
	checkCudaErrors(hipsparseSetMatType(m_desrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
	checkCudaErrors(hipsparseSetMatIndexBase(m_desrA, HIPSPARSE_INDEX_BASE_ZERO));
}

AIA_CUDAF::AIA_CUDAF(const int iM,
					 const int icols,
					 const int irows)
	: m_M(iM)
	, m_cols(icols)
	, m_rows(irows)
{
	m_N = icols * irows;

	// Allocate required pinned host memory
	m_h_old_delta = (float*)malloc(sizeof(float)*m_M);
	WFT_FPA::Utils::cucreateptr(m_h_delta, sizeof(float)*m_M);
	WFT_FPA::Utils::cucreateptr(m_h_A2, sizeof(float) * 9);
	WFT_FPA::Utils::cucreateptr(m_h_b2, sizeof(float)*m_M * 3);

	// Allocate device memory
	checkCudaErrors(hipMalloc((void**)&m_d_csrValA1, sizeof(float) * 9 * m_N));
	checkCudaErrors(hipMalloc((void**)&m_d_csrRowPtrA1, sizeof(int)*(3 * m_N + 1)));
	checkCudaErrors(hipMalloc((void**)&m_d_csrColIndA1, sizeof(int) * 9 * m_N));
	checkCudaErrors(hipMalloc((void**)&m_d_b1, sizeof(float) * 3 * m_N));
	checkCudaErrors(hipMalloc((void**)&m_d_phi, sizeof(float) * m_cols * m_rows));
	checkCudaErrors(hipMalloc((void**)&m_d_delta, sizeof(float) * m_M));
	checkCudaErrors(hipMalloc((void**)&m_d_img, sizeof(uchar)*m_M*m_N));

	// Initialize the csrRowPtrA & csrColIndA here because they remain at the same patterns
	generate_csrColIndA1_csrRowPtrA1_kernel<<<8*32, 256>>>(m_d_csrColIndA1, m_d_csrRowPtrA1, m_N);
	getLastCudaError("generate_csrColIndA1_csrRowPtrA1_kernel launch failed!");

	// Create cuSolver required handles
	checkCudaErrors(hipsolverSpCreate(&m_cuSolverHandle));
	checkCudaErrors(hipStreamCreate(&m_stream));
	checkCudaErrors(hipsolverSpSetStream(m_cuSolverHandle, m_stream));
	checkCudaErrors(hipsparseCreateMatDescr(&m_desrA));
	checkCudaErrors(hipsparseSetMatType(m_desrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
	checkCudaErrors(hipsparseSetMatIndexBase(m_desrA, HIPSPARSE_INDEX_BASE_ZERO));
}

AIA_CUDAF::~AIA_CUDAF()
{
	checkCudaErrors(hipsolverSpDestroy(m_cuSolverHandle));
	checkCudaErrors(hipStreamDestroy(m_stream));
	checkCudaErrors(hipsparseDestroyMatDescr(m_desrA));

	WFT_FPA::Utils::cudestroyptr(m_h_A2);
	WFT_FPA::Utils::cudestroyptr(m_h_b2);
	WFT_FPA::Utils::cudestroyptr(m_h_delta);

	free(m_h_old_delta);	m_h_old_delta = nullptr;
	WFT_FPA::Utils::cudaSafeFree(m_d_csrValA1);
	WFT_FPA::Utils::cudaSafeFree(m_d_csrColIndA1);
	WFT_FPA::Utils::cudaSafeFree(m_d_csrRowPtrA1);
	WFT_FPA::Utils::cudaSafeFree(m_d_b1);
	WFT_FPA::Utils::cudaSafeFree(m_d_phi);
	WFT_FPA::Utils::cudaSafeFree(m_d_delta);
}

void AIA_CUDAF::operator() (//Outputs
			 				std::vector<float>& v_phi,
							std::vector<float>& v_deltas,
							double &runningtime,
							int &iters,
							float &err,
							// Inputs
							const std::vector<cv::Mat>& v_f,
							int iMaxIterations,
							float dMaxErr,
							int iNumThreads)
{
	omp_set_num_threads(iNumThreads);

	/* If the number of frames != the number of initial deltas, randomly generate 
	   deltas */	
	if (m_M != v_deltas.size())
	{
		// Mimic the real randomness by varying the seed
		std::default_random_engine generator;
		generator.seed(time(nullptr));

		// Random numbers picked from standard normal distribution g(0,1)
		std::normal_distribution<float> distribution(0.0f, 1.0f);

		for (int i = 0; i < m_M; i++)
		{
			m_h_delta[i] = distribution(generator);
		}
	}
	else
	{
		for (int i = 0; i < m_M; i++)
		{
			m_h_delta[i] = v_deltas[i];
		}
	}

	// Copy the images to device
	for (int i = 0; i < m_M; i++)
	{
		checkCudaErrors(hipMemcpy(&m_d_img[i*m_M], v_f[i].data, sizeof(uchar)*m_cols*m_rows, hipMemcpyHostToDevice));
	}
	
	err = dMaxErr * 2.0f;
	iters = 0;

	double start = omp_get_wtime();



	// Generate csrValA & RHS
	generate_csrValA1_rhs1_kernel<<<8*32, 256>>>(m_d_csrValA1, m_d_b1, m_d_img, m_d_delta, m_M, m_N);
	getLastCudaError("generate_csrValA1_rhs1_kernel launch failed!");

	float *csrValA = (float*)malloc(sizeof(float) * 9 * m_N);
	int *csrRowPtrA = (int*)malloc(sizeof(int) * (3 * m_N + 1));
	int *csrColIndA = (int*)malloc(sizeof(int) * 9 * m_N);

	checkCudaErrors(hipMemcpy(csrValA, m_d_csrValA1, sizeof(float) * 9 * m_N, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(csrColIndA, m_d_csrColIndA1, sizeof(int) * 9 * m_N, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(csrRowPtrA, m_d_csrRowPtrA1, sizeof(int) *(3 * m_N + 1), hipMemcpyDeviceToHost));
	int issym = 0;
	checkCudaErrors(cusolverSpXcsrissymHost(m_cuSolverHandle, 3 * m_N, 9 * m_N, m_desrA, csrRowPtrA, csrRowPtrA + 1, csrColIndA, &issym));

	if (!issym)
	{
		printf("Error: A has no symmetric pattern, please use LU or QR \n");
		exit(EXIT_FAILURE);
	}

	free(csrValA);
	free(csrRowPtrA);
	free(csrColIndA);

	/* Begin the real algorithm */
	//while (err > dMaxErr && iters < iMaxIterations)
	//{
	//	for(int i=0; i<m_M; i++)
	//	{
	//		m_h_old_delta[i] = m_h_delta[i];
	//	}

	//	// Step 1: pixel-by-pixel iterations
	//	computePhi();

	//	// Step 2: frame-by-frame iterations
	//	computeDelta();

	//	// Step 3: update & check convergence criterion
	//	iters++;
	//	err = computeMaxError(m_h_delta, m_h_old_delta, m_M);
	//}

}

void AIA_CUDAF::computePhi()
{
	
}

void AIA_CUDAF::computeDelta()
{}

float AIA_CUDAF::computeMaxError(const float *v_delta,
								 const float *v_deltaOld,
								 int m)
{
	std::vector<float> abs;

	for (int i = 0; i < m; i++)
	{
		abs.push_back(std::abs(v_delta[i] - v_deltaOld[i]));
	}
	std::sort(abs.begin(), abs.end(), std::greater<float>());

	return abs[0];
}

}	//	namespace AIA