#include "hip/hip_runtime.h"
#include "aia_cudaf_yctest.h"
#include <time.h>
#include <functional>
#include <memory>
#include <random>
#include <omp.h>
#include <mkl.h>
#include <fstream>
#include "hip/hip_runtime.h"
#include "hipsparse.h"
#include "hipsolver.h"
#include "hip/hip_runtime_api.h"
#include ""
#include "hip/device_functions.h"

namespace AIA {

	__inline__ __device__
		float warpReduceSum(float val) {
		for (int offset = warpSize / 2; offset > 0; offset /= 2)
			val += __shfl_down(val, offset);
		return val;
	}

	/*---------------------------------------CUDA Kernels----------------------------------*/
	__global__
	void Gaussian_Elimination_3x3_kernel_YC(const float *in_A,
		float *out_b,
		int iSize)
	{
		float A[3][4];	// The augmented matrix

		for (int i = threadIdx.x + blockDim.x * blockIdx.x;
			i < iSize;
			i += blockDim.x * gridDim.x)
		{
			int idA = i * 9; // Index in Mat A
			int idb = i * 3; // Index in Vec b

			// Load values from A&b to the augmented matrix A per thread
			A[0][0] = in_A[idA + 0];		A[0][1] = in_A[idA + 3];	A[0][2] = in_A[idA + 6];	A[0][3] = out_b[idb + 0];
			A[1][0] = A[0][1];	A[1][1] = in_A[idA + 4];	A[1][2] = in_A[idA + 7];	A[1][3] = out_b[idb + 1];
			A[2][0] = A[0][2];	A[2][1] = A[1][2];			A[2][2] = in_A[idA + 8];	A[2][3] = out_b[idb + 2];

			// Gaussian Elimination with partial pivoting algorithm
			for (int k = 0; k < 3; k++)
			{
				// 1. Find the i-th pivot of the following A[k][i] elements
				int i_max = -1;
				float i_pivot = 0.0f;

				for (int i = k; i < 3; i++)
				{
					if (fabsf(i_pivot) - fabsf(A[i][k]) <= 1e-6)
					{
						i_pivot = A[i][k];
						i_max = i;
					}

				}

				// 2. swap rows
				for (int j = 0; j < 4; j++)
				{
					float temp = A[i_max][j];
					A[i_max][j] = A[k][j];
					A[k][j] = temp;
				}

				// 3. Triangulate the matrix
				for (int i = k + 1; i < 3; i++)
				{
					float mult = A[i][k] / A[k][k];

					for (int j = 0; j < 4; j++)
					{
						A[i][j] = A[i][j] - A[k][j] * mult;
					}
				}
			}

			// 4. Find the solution using backward substitution method
			A[2][3] = A[2][3] / A[2][2];
			A[1][3] = (A[1][3] - A[2][3] * A[1][2]) / A[1][1];
			A[0][3] = (A[0][3] - A[2][3] * A[0][2] - A[1][3] * A[0][1]) / A[0][0];

			// 5. Wirte the results back to out_b
			out_b[idb + 0] = A[0][3];
			out_b[idb + 1] = A[1][3];
			out_b[idb + 2] = A[2][3];
		}
	}



	__global__
		void generate_csrValA1_rhs1_kernel_YC(float* d_out_csrValA1,
			float* d_out_csr_rhs1,
			uchar* d_in_img,
			float* d_in_delta,
			const int M,
			const int N)
	{
		/*float a3 = 0, a4 = 0, a6 = 0, a7 = 0, a8 = 0;
		for (int i = 0; i < M; i++)
		{
			float delta = d_in_delta[i];
			float cos_delta = cos(delta);
			float sin_delta = sin(delta);
		

			a3 += cos_delta;
			a4 += cos_delta * cos_delta;
			a6 += sin_delta;
			a7 += sin_delta * cos_delta;
			a8 += sin_delta * sin_delta;

		}

		for (int j = threadIdx.x + blockDim.x *blockIdx.x;
			j < N;
			j += blockDim.x * gridDim.x)
		{
			
			float b0 = 0, b1 = 0, b2 = 0;

			for (int i = 0; i < M; i++)
			{
				float delta = d_in_delta[i];
				float cos_delta = cos(delta);
				float sin_delta = sin(delta);
				float Iij = static_cast<float>(d_in_img[i*N + j]);

			
				b0 += Iij;
				b1 += Iij * cos_delta;
				b2 += Iij * sin_delta;
			}

			
			d_out_csrValA1[j * 9 + 0] = M;
			d_out_csrValA1[j * 9 + 1] = a3;
			d_out_csrValA1[j * 9 + 2] = a6;
			d_out_csrValA1[j * 9 + 3] = a3;
			d_out_csrValA1[j * 9 + 4] = a4;
			d_out_csrValA1[j * 9 + 5] = a7;
			d_out_csrValA1[j * 9 + 6] = a6;
			d_out_csrValA1[j * 9 + 7] = a7;
			d_out_csrValA1[j * 9 + 8] = a8;

			d_out_csr_rhs1[j * 3 + 0] = b0;
			d_out_csr_rhs1[j * 3 + 1] = b1;
			d_out_csr_rhs1[j * 3 + 2] = b2;
		}*/
		
		for (int j = threadIdx.x + blockDim.x *blockIdx.x;
			j < N;
			j += blockDim.x * gridDim.x)
		{
			float a3 = 0, a4 = 0, a6 = 0, a7 = 0, a8 = 0;
			float b0 = 0, b1 = 0, b2 = 0;

			for (int i = 0; i < M; i++)
			{
				float delta = d_in_delta[i];
				float cos_delta = cos(delta);
				float sin_delta = sin(delta);
				float Iij = static_cast<float>(d_in_img[i*N + j]);

				a3 += cos_delta;
				a4 += cos_delta * cos_delta;
				a6 += sin_delta;
				a7 += sin_delta * cos_delta;
				a8 += sin_delta * sin_delta;
				b0 += Iij;
				b1 += Iij * cos_delta;
				b2 += Iij * sin_delta;
			}

			d_out_csrValA1[j * 9 + 0] = M;
			d_out_csrValA1[j * 9 + 1] = a3;
			d_out_csrValA1[j * 9 + 2] = a6;
			d_out_csrValA1[j * 9 + 3] = a3;
			d_out_csrValA1[j * 9 + 4] = a4;
			d_out_csrValA1[j * 9 + 5] = a7;
			d_out_csrValA1[j * 9 + 6] = a6;
			d_out_csrValA1[j * 9 + 7] = a7;
			d_out_csrValA1[j * 9 + 8] = a8;

			d_out_csr_rhs1[j * 3 + 0] = b0;
			d_out_csr_rhs1[j * 3 + 1] = b1;
			d_out_csr_rhs1[j * 3 + 2] = b2;
		}
	}

	__global__
		void generate_csrColIndA1_csrRowPtrA1_kernel_YC(int* d_out_csrColIndA1,
			int* d_out_csrRowPtrA1,
			const int N)
	{
		for (int i = threadIdx.x + blockIdx.x * blockDim.x;
			i < N;
			i += blockDim.x * gridDim.x)
		{
			int a1 = i * 9;

			d_out_csrRowPtrA1[i * 3 + 0] = a1;
			d_out_csrRowPtrA1[i * 3 + 1] = a1 + 3;
			d_out_csrRowPtrA1[i * 3 + 2] = a1 + 6;

			a1 = i * 3;
			int a2 = a1 + 1;
			int a3 = a1 + 2;

			d_out_csrColIndA1[i * 9 + 0] = a1;
			d_out_csrColIndA1[i * 9 + 1] = a2;
			d_out_csrColIndA1[i * 9 + 2] = a3;
			d_out_csrColIndA1[i * 9 + 3] = a1;
			d_out_csrColIndA1[i * 9 + 4] = a2;
			d_out_csrColIndA1[i * 9 + 5] = a3;
			d_out_csrColIndA1[i * 9 + 6] = a1;
			d_out_csrColIndA1[i * 9 + 7] = a2;
			d_out_csrColIndA1[i * 9 + 8] = a3;
		}

		// Last ele of csrRowIndA is nnz + csrRowIndA(0)
		if (blockIdx.x == 0 && threadIdx.x == 0)
			d_out_csrRowPtrA1[3 * N] = 9 * N;
	}

	__global__
		void get_phi_kernel_YC(float *d_out_phi, float* d_in_x, int N)
	{
		for (int i = threadIdx.x + blockIdx.x * blockDim.x;
			i < N;
			i += gridDim.x * blockDim.x)
		{
			d_out_phi[i] = atan2f(-d_in_x[i * 3 + 2], d_in_x[i * 3 + 1]);
		}
	}

	__global__
		void get_final_phi_kernel_YC(float* d_out_phi, float* d_in_delta, int N)
	{
		float delta0 = d_in_delta[0];

		for (int i = threadIdx.x + blockIdx.x * blockDim.x;
			i < N;
			i += gridDim.x * blockDim.x)
		{
			float temp = d_out_phi[i] + delta0;
			d_out_phi[i] = atan2f(sin(temp), cos(temp));
		}
	}

	__global__
		void generate_A2_kernel_YC(float *d_out_A2temp, int N, float* d_in_phi)
	{
		if (blockIdx.x == 0 && threadIdx.x < 5)
		{
			d_out_A2temp[threadIdx.x] = 0;
		}

		float sum1 = 0, sum2 = 0, sum3 = 0, sum4 = 0, sum5 = 0;

		for (int j = threadIdx.x + blockIdx.x *blockDim.x;
			j < N;
			j += gridDim.x * blockDim.x)
		{
			float cos_phi = cos(d_in_phi[j]);
			float sin_phi = sin(d_in_phi[j]);

			sum1 += cos_phi;
			sum2 += cos_phi * cos_phi;
			sum3 += sin_phi;
			sum4 += cos_phi * sin_phi;
			sum5 += sin_phi * sin_phi;
		}

		sum1 = warpReduceSum(sum1);
		sum2 = warpReduceSum(sum2);
		sum3 = warpReduceSum(sum3);
		sum4 = warpReduceSum(sum4);
		sum5 = warpReduceSum(sum5);

		if (threadIdx.x % warpSize == 0)
		{
			atomicAdd(&d_out_A2temp[0], sum1);
			atomicAdd(&d_out_A2temp[1], sum2);
			atomicAdd(&d_out_A2temp[2], sum3);
			atomicAdd(&d_out_A2temp[3], sum4);
			atomicAdd(&d_out_A2temp[4], sum5);
		}
	}
	__global__
		void generate_b2_kernel_YC(float *d_out_b2, int i, int N, float* d_in_phi, uchar* d_in_img)
	{
		float b1 = 0, b2 = 0, b3 = 0;

		if (blockIdx.x == 0 && threadIdx.x < 3)
		{
			d_out_b2[i * 3 + threadIdx.x] = 0;
		}

		for (int j = threadIdx.x + blockIdx.x *blockDim.x;
			j < N;
			j += gridDim.x * blockDim.x)
		{
			float Iij = static_cast<float>(d_in_img[i*N + j]);
			float cos_phi = cos(d_in_phi[j]);
			float sin_phi = sin(d_in_phi[j]);

			b1 += Iij;
			b2 += Iij * cos_phi;
			b3 += Iij * sin_phi;
		}

		b1 = warpReduceSum(b1);
		b2 = warpReduceSum(b2);
		b3 = warpReduceSum(b3);

		if (threadIdx.x % warpSize == 0)
		{
			atomicAdd(&d_out_b2[i * 3 + 0], b1);
			atomicAdd(&d_out_b2[i * 3 + 1], b2);
			atomicAdd(&d_out_b2[i * 3 + 2], b3);
		}
	}

	/*--------------------------------------End CUDA Kernels--------------------------------*/

	AIA_CUDAF_YCTEST::AIA_CUDAF_YCTEST(const std::vector<cv::Mat>& v_f)
		: m_d_img(nullptr)
		, m_d_csrColIndA1(nullptr)
		, m_d_csrValA1(nullptr)
		, m_d_csrRowPtrA1(nullptr)
		, m_d_b1(nullptr)
		, m_d_phi(nullptr)
		, m_d_delta(nullptr)
		, m_h_delta(nullptr)
		, m_h_A2(9, 0)
		, m_h_b2(nullptr)
	{
		// Get params
		m_M = v_f.size();
		m_N = v_f[0].cols*v_f[0].rows;
		m_cols = v_f[0].cols;
		m_rows = v_f[0].rows;

		// Allocate required pinned host memory
		m_h_old_delta = (float*)malloc(sizeof(float)*m_M);
		WFT_FPA::Utils::hcreateptr(m_h_delta, sizeof(float)*m_M);
		WFT_FPA::Utils::hcreateptr(m_h_A2temp, sizeof(float) * 5);
		WFT_FPA::Utils::hcreateptr(m_h_b2, sizeof(float)*m_M * 3);

		// Allocate device memory
		checkCudaErrors(hipMalloc((void**)&m_d_csrValA1, sizeof(float) * 9 * m_N));
		checkCudaErrors(hipMalloc((void**)&m_d_csrRowPtrA1, sizeof(int)*(3 * m_N + 1)));
		checkCudaErrors(hipMalloc((void**)&m_d_csrColIndA1, sizeof(int) * 9 * m_N));
		checkCudaErrors(hipMalloc((void**)&m_d_b1, sizeof(float) * 3 * m_N));
		checkCudaErrors(hipMalloc((void**)&m_d_phi, sizeof(float) * m_cols * m_rows));
		checkCudaErrors(hipMalloc((void**)&m_d_delta, sizeof(float) * m_M));
		checkCudaErrors(hipMalloc((void**)&m_d_img, sizeof(uchar)*m_M*m_N));
		checkCudaErrors(hipMalloc((void**)&m_d_b2, sizeof(float)*m_M * 3));
		checkCudaErrors(hipMalloc((void**)&m_d_A2temp, sizeof(float) * 5));

		// Initialize the csrRowPtrA & csrColIndA here because they remain at the same patterns
		generate_csrColIndA1_csrRowPtrA1_kernel_YC <<<8 * 32, 256 >>> (m_d_csrColIndA1, m_d_csrRowPtrA1, m_N);
		getLastCudaError("generate_csrColIndA1_csrRowPtrA1_kernel launch failed!");

		// Create cuSolver required handles
		checkCudaErrors(hipsolverSpCreate(&m_cuSolverHandle));
		checkCudaErrors(hipsparseCreateMatDescr(&m_desrA));
		checkCudaErrors(hipsparseSetMatType(m_desrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
		checkCudaErrors(hipsparseSetMatIndexBase(m_desrA, HIPSPARSE_INDEX_BASE_ZERO));
	}

	AIA_CUDAF_YCTEST::AIA_CUDAF_YCTEST(const int iM,
		const int icols,
		const int irows)
		: m_M(iM)
		, m_cols(icols)
		, m_rows(irows)
		, m_h_A2(9, 0)
	{
		m_N = icols * irows;

		// Allocate required pinned host memory
		m_h_old_delta = (float*)malloc(sizeof(float)*m_M);
		WFT_FPA::Utils::hcreateptr(m_h_delta, sizeof(float)*m_M);
		WFT_FPA::Utils::hcreateptr(m_h_A2temp, sizeof(float) * 5);
		WFT_FPA::Utils::hcreateptr(m_h_b2, sizeof(float)*m_M * 3);

		// Allocate device memory
		checkCudaErrors(hipMalloc((void**)&m_d_csrValA1, sizeof(float) * 9 * m_N));
		checkCudaErrors(hipMalloc((void**)&m_d_csrRowPtrA1, sizeof(int)*(3 * m_N + 1)));
		checkCudaErrors(hipMalloc((void**)&m_d_csrColIndA1, sizeof(int) * 9 * m_N));
		checkCudaErrors(hipMalloc((void**)&m_d_b1, sizeof(float) * 3 * m_N));
		checkCudaErrors(hipMalloc((void**)&m_d_phi, sizeof(float) * m_cols * m_rows));
		checkCudaErrors(hipMalloc((void**)&m_d_delta, sizeof(float) * m_M));
		checkCudaErrors(hipMalloc((void**)&m_d_img, sizeof(uchar)*m_M*m_N));
		checkCudaErrors(hipMalloc((void**)&m_d_b2, sizeof(float)*m_M * 3));
		checkCudaErrors(hipMalloc((void**)&m_d_A2temp, sizeof(float) * 5));

		// Initialize the csrRowPtrA & csrColIndA here because they remain at the same patterns
		generate_csrColIndA1_csrRowPtrA1_kernel_YC <<<8 * 32, 256 >>> (m_d_csrColIndA1, m_d_csrRowPtrA1, m_N);
		getLastCudaError("generate_csrColIndA1_csrRowPtrA1_kernel launch failed!");

		// Create cuSolver required handles
		checkCudaErrors(hipsolverSpCreate(&m_cuSolverHandle));
		checkCudaErrors(hipsparseCreateMatDescr(&m_desrA));
		checkCudaErrors(hipsparseSetMatType(m_desrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
		checkCudaErrors(hipsparseSetMatIndexBase(m_desrA, HIPSPARSE_INDEX_BASE_ZERO));
	}

	AIA_CUDAF_YCTEST::~AIA_CUDAF_YCTEST()
	{
		checkCudaErrors(hipsolverSpDestroy(m_cuSolverHandle));
		checkCudaErrors(hipsparseDestroyMatDescr(m_desrA));

		WFT_FPA::Utils::hdestroyptr(m_h_A2temp);
		WFT_FPA::Utils::hdestroyptr(m_h_b2);
		WFT_FPA::Utils::hdestroyptr(m_h_delta);

		free(m_h_old_delta);	m_h_old_delta = nullptr;
		WFT_FPA::Utils::cudaSafeFree(m_d_csrValA1);
		WFT_FPA::Utils::cudaSafeFree(m_d_csrColIndA1);
		WFT_FPA::Utils::cudaSafeFree(m_d_csrRowPtrA1);
		WFT_FPA::Utils::cudaSafeFree(m_d_b1);
		WFT_FPA::Utils::cudaSafeFree(m_d_phi);
		WFT_FPA::Utils::cudaSafeFree(m_d_delta);
		WFT_FPA::Utils::cudaSafeFree(m_d_A2temp);
		WFT_FPA::Utils::cudaSafeFree(m_d_b2);
	}

	void AIA_CUDAF_YCTEST::operator() (//Outputs
		std::vector<float>& v_phi,
		std::vector<float>& v_deltas,
		double &runningtime,
		int &iters,
		float &err,
		// Inputs
		const std::vector<cv::Mat>& v_f,
		int iMaxIterations,
		float dMaxErr,
		int iNumThreads)
	{
		omp_set_num_threads(iNumThreads);

		/* If the number of frames != the number of initial deltas, randomly generate
		   deltas */
		if (m_M != v_deltas.size())
		{
			// Mimic the real randomness by varying the seed
			std::default_random_engine generator;
			generator.seed(time(nullptr));

			// Random numbers picked from standard normal distribution g(0,1)
			std::normal_distribution<float> distribution(0.0f, 1.0f);

			for (int i = 0; i < m_M; i++)
			{
				m_h_delta[i] = distribution(generator);
			}
		}
		else
		{
			for (int i = 0; i < m_M; i++)
			{
				m_h_delta[i] = v_deltas[i];
			}
		}

		// Copy the images to device
		for (int i = 0; i < m_M; i++)
		{
			checkCudaErrors(hipMemcpy(&m_d_img[i*m_N], v_f[i].data, sizeof(uchar)*m_N, hipMemcpyHostToDevice));
		}

		err = dMaxErr * 2.0f;
		iters = 0;


		double start = omp_get_wtime();

		/* Begin the real algorithm */
		while (err > dMaxErr && iters < iMaxIterations)
		{
			for (int i = 0; i < m_M; i++)
			{
				m_h_old_delta[i] = m_h_delta[i];
			}

			// Step 1: pixel-by-pixel iterations
			
			computePhi_YC();

			// Step 2: frame-by-frame iterations
			computeDelta_YC();

			// Step 3: update & check convergence criterion
			iters++;
			err = computeMaxError_YC(m_h_delta, m_h_old_delta, m_M);
		}

		double end = omp_get_wtime();
		runningtime = 1000.0f*(end - start);


		/* One more round to calculate phi once delta is good enough */
		computePhi_YC();

		/* Get the final phi and  deltas */
		get_final_phi_kernel_YC <<<8 * 32, 256 >>> (m_d_phi, m_d_delta, m_N);
		getLastCudaError("get_final_phi_kernel launch failed!");

		v_phi.resize(m_N);

		hipMemcpy(v_phi.data(), m_d_phi, sizeof(float)*m_N, hipMemcpyDeviceToHost);

		v_deltas.resize(m_M);

		for (int i = m_M - 1; i >= 0; i--)
		{
			m_h_delta[i] -= m_h_delta[0];
			m_h_delta[i] = atan2(sin(m_h_delta[i]), cos(m_h_delta[i]));
			v_deltas[i] = m_h_delta[i];
		}
	}

	void AIA_CUDAF_YCTEST::computePhi_YC()
	{
	
		// Load the new deltas
		checkCudaErrors(hipMemcpy(m_d_delta, m_h_delta, sizeof(float)*m_M, hipMemcpyHostToDevice));

		/*hipEvent_t start, end;
		hipEventCreate(&start);
		hipEventCreate(&end);
		hipEventRecord(start);*/

		// Generate csrValA & RHS
		generate_csrValA1_rhs1_kernel_YC <<<8 * 32, 256 >>> (m_d_csrValA1, m_d_b1, m_d_img, m_d_delta, m_M, m_N);
		getLastCudaError("generate_csrValA1_rhs1_kernel_YC launch failed!");

		//std::vector<float> h_b(m_N*3);
		//hipMemcpy(h_b.data(), m_d_b1, sizeof(float) * 3 * m_N, hipMemcpyDeviceToHost);

		Gaussian_Elimination_3x3_kernel_YC <<<256, 256 >>> (m_d_csrValA1, m_d_b1, m_N);
		getLastCudaError("Gaussian_Elimination_3x3_kernel_YC launch failed!");

		/*std::vector<float>h_b1(m_N * 3);
		hipMemcpy(h_b1.data(), m_d_b1, sizeof(float)*m_N * 3, hipMemcpyDeviceToHost);
		std::ofstream out("Phi_GPU.csv", std::ios::out | std::ios::trunc);

		for (int i = 0; i < 256 * 3 * 256; i++)
		{

			out << h_b1[i];

			out << "\n";
		}
		out.close();*/

		/*float tol = 1e-4f;
		int singularity = -1;
		checkCudaErrors(hipsolverSpScsrlsvchol(m_cuSolverHandle, 3 * m_N, 9 * m_N, m_desrA,
			m_d_csrValA1, m_d_csrRowPtrA1, m_d_csrColIndA1, m_d_b1,
			tol, 0, m_d_b1, &singularity));
		if (0 <= singularity)
		{
			printf("WARNING: the matrix is singular at row %d under tol (%E)\n", singularity, tol);
		}*/

		// Update phi
		get_phi_kernel_YC <<<8 * 32, 256 >>> (m_d_phi, m_d_b1, m_N);
		getLastCudaError("get_phi_kernel launch failed!");

		/*hipEventRecord(end);
		hipEventSynchronize(end);
		float t = 0;
		//hipEventElapsedTime(&t, start, end);*/
		//std::vector<float>h_phi(m_N);
		//hipMemcpy(h_phi.data(), m_d_phi, sizeof(float)*m_N, hipMemcpyDeviceToHost);


	}

	void AIA_CUDAF_YCTEST::computeDelta_YC()
	{
		// Generate A2
		generate_A2_kernel_YC <<<8 * 32, 256 >>> (m_d_A2temp, m_N, m_d_phi);
		getLastCudaError("generate_A2_kernel launch failed!");

		checkCudaErrors(hipMemcpy(m_h_A2temp, m_d_A2temp, sizeof(float) * 5, hipMemcpyDeviceToHost));

		// Generate b2
		for (int i = 0; i < m_M; i++)
		{
			generate_b2_kernel_YC <<<8 * 32, 256 >>> (m_d_b2, i, m_N, m_d_phi, m_d_img);
			getLastCudaError("generate_b2_kernel launch failed!");
		}

		checkCudaErrors(hipMemcpy(m_h_b2, m_d_b2, sizeof(float) * 3 * m_M, hipMemcpyDeviceToHost));

		//hipDeviceSynchronize();

		m_h_A2[0] = float(m_N);		m_h_A2[1] = m_h_A2temp[0];	m_h_A2[2] = m_h_A2temp[2];
		m_h_A2[3] = m_h_A2temp[0];	m_h_A2[4] = m_h_A2temp[1];	m_h_A2[5] = m_h_A2temp[3];
		m_h_A2[6] = m_h_A2temp[2];	m_h_A2[7] = m_h_A2temp[3];	m_h_A2[8] = m_h_A2temp[4];


		/*std::cout << "A2: " << std::endl;

		for(int i=0; i<3;i++)
		{
			for(int j=0;j<3;j++)
			{
				std::cout<<m_h_A2[i*3+j]<<",";
			}
			std::cout << std::endl;
		}*/

		/* Solve the Ax = b */
		int info = LAPACKE_sposv(LAPACK_COL_MAJOR, 'U', 3, m_M, m_h_A2.data(), 3, m_h_b2, 3);
		/* Check for the positive definiteness */
		if (info > 0) {
			printf("The leading minor of order %i is not positive ", info);
			printf("definite;\nThe solution could not be computed.\n");
			exit(1);
		}

		for (int i = 0; i < m_M; i++)
		{
			m_h_delta[i] = atan2(-m_h_b2[i * 3 + 2], m_h_b2[i * 3 + 1]);
		}
	}

	float AIA_CUDAF_YCTEST::computeMaxError_YC(const float *v_delta,
		const float *v_deltaOld,
		int m)
	{
		std::vector<float> abs;

		for (int i = 0; i < m; i++)
		{
			abs.push_back(std::abs((v_delta[i]- v_delta[0]) - (v_deltaOld[i]- v_deltaOld[0])));
		}
		std::sort(abs.begin(), abs.end(), std::greater<float>());

		return abs[0];
	}

}	//	namespace AIA
