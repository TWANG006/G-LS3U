#include "hip/hip_runtime.h"
#include "gtest\gtest.h"

#include "hip/hip_runtime.h"
#include ""
#include "hipfft/hipfft.h"
#include "hip/hip_runtime_api.h"
#define _USE_MATH_DEFINES
#include <math.h>
#include <fstream>
#include <opencv2\opencv.hpp>
#include "aia_cpuf.h"
#include "dpra_cudaf.h"
#include "cuda_testt.h"

__global__ void Gen_xf_yf_Kernel2D(
	hipfftReal *xf, hipfftReal *yf, 
	int iWidth, int iHeight)
{
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	int id = i*iWidth + j;
	int iH = iHeight / 2;
	int iW = iWidth / 2;

	if (i < iHeight && j < iWidth)
	{
		xf[id] = j - iW;
		yf[id] = i - iH;
	}
}
__global__ void Gen_xf_yf_Kernel1D(
	hipfftReal *xf, hipfftReal *yf, 
	int iWidth, int iHeight)
{
	int iH = iHeight / 2;
	int iW = iWidth / 2;

	for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < iWidth * iHeight; i += blockDim.x*gridDim.x)
	{
		int x = i % iWidth;
		int y = i / iWidth;
		
		if (y < iHeight && x < iWidth)
		{
			xf[i] = x - iW;
			yf[i] = y - iH;
		}
	}
}
__global__ void gen_xf_yf_Kernel(hipfftReal *xf, hipfftReal *yf, int iWidth, int iHeight)
{
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	int j = threadIdx.x + blockIdx.x * blockDim.x;

	int id = i*iWidth + j;

	int iHalfy = iHeight / 2;
	int iHalfx = iWidth / 2;

	if (i < iHeight && j < iWidth)
	{
		xf[id] = j - iHalfx;
		yf[id] = i - iHalfy;
	}
	
}
__global__ void shift_xf_yf_Kernel(hipfftReal *xf, hipfftReal *yf,  int iWidth, int iHeight)
{
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	int j = threadIdx.x + blockIdx.x * blockDim.x;

	int id = i*iWidth + j;

	int iHalfx = iWidth / 2;
	int iHalfy = iHeight / 2;
	int iSlice = iWidth * iHeight;

	int idQ13 = iSlice / 2 + iHalfx;
	int idQ24 = iSlice / 2 - iHalfx;

	hipfftReal Tempx, Tempy;

	if (j < iHalfx)
	{
		if(i < iHalfy)
		{
			Tempx = xf[id];
			Tempy = yf[id];

			// First Quadrant
			xf[id] = xf[id + idQ13];
			yf[id] = yf[id + idQ13];

			// Third Quadrant
			xf[id + idQ13] = Tempx;
			yf[id + idQ13] = Tempy;
		}
	}
	else
	{
		if (i < iHalfy)
		{
			Tempx = xf[id];
			Tempy = yf[id];

			// Second Quadrant
			xf[id] = xf[id + idQ24];
			yf[id] = yf[id + idQ24];

			// Fourth Quadrant
			xf[id + idQ24] = Tempx;
			yf[id + idQ24] = Tempy;
		}
	}
}
__global__
void compute_Fg_kernel(hipfftReal *d_in_xf, hipfftReal *d_in_yf, int iPaddedWidth, int iPaddedHeight, 
					   int wxt, int wyt, float wxi, float wyi, float wxl, float wyl,
					   float sigmax, float sigmay, float sn2, hipfftComplex *d_out_Fg)
{
	hipfftReal rwxt = wxl + hipfftReal(wxt) * wxi;
	hipfftReal rwyt = wyl + hipfftReal(wyt) * wyi;

	for (int i = threadIdx.x + blockIdx.x * blockDim.x;
		 i < iPaddedHeight*iPaddedWidth;
		 i += blockDim.x * gridDim.x)
	{
		hipfftReal tempx = d_in_xf[i] * 2 * hipfftReal(M_PI) * (1.0f / iPaddedWidth) - rwxt;
		hipfftReal tempy = d_in_yf[i] * 2 * hipfftReal(M_PI) * (1.0f / iPaddedHeight) - rwyt;
		tempx = -tempx * tempx * 0.5f * sigmax * sigmax;
		tempy = -tempy * tempy * 0.5f * sigmay * sigmay;
		
		d_out_Fg[i].x = exp(tempx + tempy)*sn2;
		d_out_Fg[i].y = 0;
	}
}

TEST(XF_YF_2D_1D, KernelTest)
{
	hipEvent_t start, end2D, end1D;
	hipEventCreate(&start);
	hipEventCreate(&end1D);
	hipEventCreate(&end2D);

	dim3 threads(16, 16);
	dim3 blocks((1120 + 16 - 1) / 16, (1120 + 16 - 1) / 16);

	hipfftReal *xf, *xf1, *yf, *yf1;
	checkCudaErrors(hipMalloc((void**)&xf, sizeof(hipfftReal) * 1120 * 1120));
	checkCudaErrors(hipMalloc((void**)&yf, sizeof(hipfftReal) * 1120 * 1120));
	checkCudaErrors(hipMalloc((void**)&xf1, sizeof(hipfftReal) * 1120 * 1120));
	checkCudaErrors(hipMalloc((void**)&yf1, sizeof(hipfftReal) * 1120 * 1120));

	hipEventRecord(start);
	Gen_xf_yf_Kernel1D <<<32 * 8, 256 >> >(xf1, yf1, 1120, 1120);
	getLastCudaError("1D Kernel Failed.");
	hipEventRecord(end1D);
	Gen_xf_yf_Kernel2D<<<blocks, threads>>>(xf, yf, 1120, 1120);
	getLastCudaError("2D Kernel Failed.");
	hipEventRecord(end2D);

	hipDeviceSynchronize();


	hipfftReal *hxf, *hxf1, *hyf, *hyf1;
	hxf = (hipfftReal*)malloc(sizeof(hipfftReal) * 1120 * 1120);
	hyf = (hipfftReal*)malloc(sizeof(hipfftReal) * 1120 * 1120);
	hxf1 = (hipfftReal*)malloc(sizeof(hipfftReal) * 1120 * 1120);
	hyf1 = (hipfftReal*)malloc(sizeof(hipfftReal) * 1120 * 1120);

	checkCudaErrors(hipMemcpy(hxf, xf, sizeof(hipfftReal) * 1120 * 1120, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(hyf, yf, sizeof(hipfftReal) * 1120 * 1120, hipMemcpyDeviceToHost));
	
	checkCudaErrors(hipMemcpy(hxf1, xf1, sizeof(hipfftReal) * 1120 * 1120, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(hyf1, yf1, sizeof(hipfftReal) * 1120 * 1120, hipMemcpyDeviceToHost));

	for(int i=0; i<1120; i++)
		for(int j=0; j<1120; j++)
		{
			ASSERT_EQ(hxf[i * 1120 + j], hxf1[i * 1120 + j]);
			ASSERT_EQ(hyf[i * 1120 + j], hyf1[i * 1120 + j]);
		}

	float t1, t2;
	hipEventElapsedTime(&t1, start, end1D);
	hipEventElapsedTime(&t2, end1D, end2D);

	std::cout << "1D Kernel Execution Time: " << t1 << std::endl;
	std::cout << "2D Kernel Execution Time: " << t2 << std::endl;

	hipEventDestroy(start);
	hipEventDestroy(end1D);
	hipEventDestroy(end2D);
	hipFree(xf);
	hipFree(yf);
	hipFree(xf1);
	hipFree(yf1);
	free(hxf);
	free(hyf);
	free(hxf1);
	free(hyf1);
}
TEST(XF_YF_2D_with_FFTSHIFT, KernelTest)
{
	int iWidth = 64;
	int iHeight = 50;

	hipEvent_t start, end2D;
	hipEventCreate(&start);
	hipEventCreate(&end2D);

	dim3 threads(16, 16);
	dim3 blocks((iWidth + 16 - 1) / 16, (iHeight + 16 - 1) / 16);
	dim3 blocks1((iWidth + 16 - 1) / 16, (iHeight + 16 - 1) / 16);

	hipfftReal *xf, *yf;
	checkCudaErrors(hipMalloc((void**)&xf, sizeof(hipfftReal) * iWidth * iHeight));
	checkCudaErrors(hipMalloc((void**)&yf, sizeof(hipfftReal) * iWidth * iHeight));

	hipEventRecord(start);
	gen_xf_yf_Kernel<<<blocks, threads>>>(xf, yf, iWidth, iHeight);
	getLastCudaError("2D Kernel Failed.");
	shift_xf_yf_Kernel<<<blocks1, threads>>>(xf, yf, iWidth, iHeight);
	getLastCudaError("2D Shift Failed.");
	hipEventRecord(end2D);

	hipDeviceSynchronize();


	hipfftReal *hxf, *hyf;
	hxf = (hipfftReal*)malloc(sizeof(hipfftReal) * iWidth * iHeight);
	hyf = (hipfftReal*)malloc(sizeof(hipfftReal) * iWidth * iHeight);

	checkCudaErrors(hipMemcpy(hxf, xf, sizeof(hipfftReal) * iWidth * iHeight, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(hyf, yf, sizeof(hipfftReal) * iWidth * iHeight, hipMemcpyDeviceToHost));

	std::ofstream out("xfyf.csv", std::ios::out | std::ios::trunc);

	for(int i=0; i<iHeight; i++)
	{
		for(int j=0; j<iWidth; j++)
		{
			out << "[" << hxf[i * iWidth + j] << "-" << hyf[i * iWidth + j] << "]" << ",";
		}
		out<<"\n";
	}

	out.close();

	float t1;
	hipEventElapsedTime(&t1, start, end2D);

	std::cout << "1D Kernel Execution Time: " << t1 << std::endl;

	hipEventDestroy(start);
	hipEventDestroy(end2D);
	hipFree(xf);
	hipFree(yf);

	free(hxf);
	free(hyf);

}
////TEST(Fg_Computation, KernelTest)
////{
////	int iWidth = 192;
////	int iHeight = 192;
////
////	dim3 threads(16, 16);
////	dim3 blocks((iWidth + 16 - 1) / 16, (iHeight + 16 - 1) / 16);
////	dim3 blocks1((iWidth + 16 - 1) / 16, (iHeight + 16 - 1) / 16);
////
////	hipfftReal *xf, *yf;
////	hipfftComplex *fg;
////	checkCudaErrors(hipMalloc((void**)&xf, sizeof(hipfftReal) * iWidth * iHeight));
////	checkCudaErrors(hipMalloc((void**)&yf, sizeof(hipfftReal) * iWidth * iHeight));
////	checkCudaErrors(hipMalloc((void**)&fg, sizeof(hipfftComplex) * iWidth * iHeight));
////
////	gen_xf_yf_Kernel<<<blocks, threads>>>(xf, yf, iWidth, iHeight);
////	getLastCudaError("gen_xf_yf_Kernel Kernel Failed.");
////
////	shift_xf_yf_Kernel<<<blocks1, threads>>>(xf, yf, iWidth, iHeight);
////	getLastCudaError("2D Shift Failed.");
////
////	compute_Fg_kernel<<<32*8, 256>>>(xf, yf, iWidth, iHeight, 0, 0, 0.1f, 0.1f, -2.3f, -2.3f, 10, 10, 35.4491f, fg);
////	getLastCudaError("compute_Fg_kernel Kernel Failed.");
////
////	hipfftComplex *hfg;
////	hipfftReal *hfx, *hfy;
////	hfg = (hipfftComplex*)malloc(sizeof(hipfftComplex) * iWidth * iHeight);
////	hfx = (hipfftReal*)malloc(sizeof(hipfftReal) * iWidth * iHeight);
////	hfy = (hipfftReal*)malloc(sizeof(hipfftReal) * iWidth * iHeight);
////
////	checkCudaErrors(hipMemcpy(hfg, fg, sizeof(hipfftComplex) * iWidth * iHeight, hipMemcpyDeviceToHost));
////	checkCudaErrors(hipMemcpy(hfx, xf, sizeof(hipfftReal) * iWidth * iHeight, hipMemcpyDeviceToHost));
////	checkCudaErrors(hipMemcpy(hfy, yf, sizeof(hipfftReal) * iWidth * iHeight, hipMemcpyDeviceToHost));
////
////	std::ofstream out("CUDAFg.csv", std::ios::out | std::ios::trunc);
////
////	for(int i=0; i<iHeight; i++)
////	{
////		for(int j=0; j<iWidth; j++)
////		{
////			out << hfg[i * iWidth + j].x << "+" << hfg[i * iWidth + j].y << "i" << ",";
////		}
////		out<<"\n";
////	}
////
////	out.close();
////
////	out.open("xfyf.csv", std::ios::out | std::ios::trunc);
////
////	for(int i=0; i<iHeight; i++)
////	{
////		for(int j=0; j<iWidth; j++)
////		{
////			out << "[" << hfx[i * iWidth + j] << "-" << hfy[i * iWidth + j] << "]" << ",";
////		}
////		out<<"\n";
////	}
////
////	out.close();
////
////	hipFree(xf);
////	hipFree(yf);
////	hipFree(fg);
////	free(hfg);
////	free(hfx);
////	free(hfy);
////}
//
//#define BLOCK_SIZE_16 16
//
//__global__
//void generate_csrValA_b_kernel(float *d_out_csrValA,
//							   float *d_out_b,
//							   const uchar *d_in_img,
//							   const float *d_in_phi,
//							   const int iWidth,
//							   const int iHeight)
//{
//	const int y = threadIdx.y + (BLOCK_SIZE_16 - 2) * blockIdx.y;
//	const int x = threadIdx.x + (BLOCK_SIZE_16 - 2) * blockIdx.x;
//
//	int idA = (y*iWidth + x) * 9;
//
//	float sum_cos = 0, sum_sin = 0, sum_sincos = 0, sum_sin2 = 0, sum_cos2 = 0;
//	float sum_ft = 0, sum_ft_cos = 0, sum_ft_sin = 0;
//
//	// Global Memory offset: every block actually begin with 2 overlapped pixels
//	__shared__ float phi_sh[BLOCK_SIZE_16][BLOCK_SIZE_16];
//	__shared__ uchar img_sh[BLOCK_SIZE_16][BLOCK_SIZE_16];
//
//	// Load the global mem to shared mem
//	if (y < iHeight && x < iWidth)
//	{
//		phi_sh[threadIdx.y][threadIdx.x] = d_in_phi[y*iWidth + x];
//		img_sh[threadIdx.y][threadIdx.x] = d_in_img[y*iWidth + x];
//	}
//	__syncthreads();	
//
//	if (y < iHeight && x < iWidth)
//	{
//		// Compute the results within the boundary
//		if (y >= 1 && y < iHeight - 1 && x >= 1 && x < iWidth - 1 &&
//			threadIdx.x != 0 && threadIdx.x != BLOCK_SIZE_16 - 1 &&
//			threadIdx.y != 0 && threadIdx.y != BLOCK_SIZE_16 - 1)
//		{
//			sum_cos = 0, sum_sin = 0, sum_sincos = 0, sum_sin2 = 0, sum_cos2 = 0;
//			sum_ft = 0, sum_ft_cos = 0, sum_ft_sin = 0;
//
//			for (int i = threadIdx.y - 1; i <= threadIdx.y + 1; i++)
//			{
//				for (int j = threadIdx.x - 1; j <= threadIdx.x + 1; j++)
//				{
//					float cos_phi = cos(phi_sh[i][j]);
//					float sin_phi = sin(phi_sh[i][j]);
//					float ft = static_cast<float>(img_sh[i][j]);
//
//					// Elements of A
//					sum_cos += cos_phi;
//					sum_sin += sin_phi;
//					sum_sincos += cos_phi * sin_phi;
//					sum_sin2 += sin_phi*sin_phi;
//					sum_cos2 += cos_phi*cos_phi;
//
//					// Elements of b
//					sum_ft += ft;
//					sum_ft_cos += ft * cos_phi;
//					sum_ft_sin += ft * sin_phi;
//				}
//			}
//			d_out_csrValA[idA + 0] = 9;			d_out_csrValA[idA + 1] = sum_cos;		d_out_csrValA[idA + 2] = sum_sin;
//			d_out_csrValA[idA + 3] = sum_cos;	d_out_csrValA[idA + 4] = sum_cos2;		d_out_csrValA[idA + 5] = sum_sincos;
//			d_out_csrValA[idA + 6] = sum_sin;	d_out_csrValA[idA + 7] = sum_sincos;	d_out_csrValA[idA + 8] = sum_sin2;
//		}
//		// Deal with boundary
//		if ((y == 0 && blockIdx.y == 0) ||
//		    (x == 0 && blockIdx.x == 0) ||
//			(y == iHeight - 1 && blockIdx.y == gridDim.y - 1) ||
//			(x == iWidth - 1 && blockIdx.x == gridDim.x - 1))
//		{
//			sum_cos = 0, sum_sin = 0, sum_sincos = 0, sum_sin2 = 0, sum_cos2 = 0;
//			sum_ft = 0, sum_ft_cos = 0, sum_ft_sin = 0;
//
//			int yl = -1, yh = 1, xl = -1, xh = 1;
//
//			if (y == 0)				
//			{
//				yl = 0;	yh = 1;
//			}
//			if (y == iHeight - 1)	
//			{
//				yl = -1;	yh = 0;
//			}
//			if (x == 0)		
//			{
//				xl = 0;	xh = 1;
//			}
//			if (x == iWidth - 1)
//			{
//				xl = -1;	xh = 0;
//			}
//
//			for (int i = yl; i <= yh; i++)
//			{
//				for (int j = xl; j <= xh; j++)
//				{
//					float cos_phi = cos(phi_sh[threadIdx.y + i][threadIdx.x + j]);
//					float sin_phi = sin(phi_sh[threadIdx.y + i][threadIdx.x + j]);
//					float ft = static_cast<float>(img_sh[threadIdx.y + i][threadIdx.x + j]);
//
//					// Elements of A
//					sum_cos += cos_phi;
//					sum_sin += sin_phi;
//					sum_sincos += cos_phi * sin_phi;
//					sum_sin2 += sin_phi*sin_phi;
//					sum_cos2 += cos_phi*cos_phi;
//
//					// Elements of b
//					sum_ft += ft;
//					sum_ft_cos += ft * cos_phi;
//					sum_ft_sin += ft * sin_phi;
//				}
//			}
//			d_out_csrValA[idA + 0] = 9;			d_out_csrValA[idA + 1] = sum_cos;		d_out_csrValA[idA + 2] = sum_sin;
//			d_out_csrValA[idA + 3] = sum_cos;	d_out_csrValA[idA + 4] = sum_cos2;		d_out_csrValA[idA + 5] = sum_sincos;
//			d_out_csrValA[idA + 6] = sum_sin;	d_out_csrValA[idA + 7] = sum_sincos;	d_out_csrValA[idA + 8] = sum_sin2;
//		}
//	}
//}
//
//void launch()
//{
//		/* AIA to get the initial phi */
//	std::vector<cv::Mat> f;
//
//	cv::Mat img = cv::imread("1.bmp");
//	cv::cvtColor(img,
//				 img,
//				 CV_BGR2GRAY);
//	f.push_back(img);
//
//	img = cv::imread("2.bmp");
//	cv::cvtColor(img,
//				 img,
//				 CV_BGR2GRAY);
//	f.push_back(img);
//
//	img = cv::imread("3.bmp");
//	cv::cvtColor(img,
//				 img,
//				 CV_BGR2GRAY);
//	f.push_back(img);
//	
//
//	// computation
//	std::vector<float> phi;
//	std::vector<float> delta{-1.4577f,
//   -0.8285f,
//    1.3368f};
//	double time = 0;
//	float err = 0;
//	int iter = 0;
//
//	AIA::AIA_CPU_DnF aia;
//	aia(phi, delta, time, iter, err, f, 20, 1e-4, 12);
//
//	std::cout << "AIA Running Time: " << time << std::endl;
//	std::cout << "AIA Error is: " << err << std::endl;
//	std::cout << "AIA Iteration is: " << iter << std::endl;
//
//	std::cout << "Delta is: " << delta[0] << "," << delta[1] << "," << delta[2] << std::endl;
//
//	int iSize = f[0].cols * f[0].rows;
//
//	float *m_d_csrValA, *m_d_b;
//
//
//	float *dPhi0, *dphi_curr = nullptr;
//	uchar *dImg;
//	hipMalloc((void**)&dPhi0, sizeof(float)*f[0].cols*f[0].rows);
//	hipMalloc((void**)&dImg, sizeof(uchar)*f[0].cols*f[0].rows);
//	hipMemcpy(dPhi0, phi.data(), sizeof(float)*f[0].cols*f[0].rows, hipMemcpyHostToDevice);
//	hipMemcpy(dImg, f[0].data, sizeof(uchar)*f[0].cols*f[0].rows, hipMemcpyHostToDevice);
//	checkCudaErrors(hipMalloc((void**)&m_d_csrValA, sizeof(float) * 9 * iSize));
//	checkCudaErrors(hipMalloc((void**)&m_d_b, sizeof(float) * 3 * iSize));
//
//
//	dim3 threads(BLOCK_SIZE_16, BLOCK_SIZE_16);
//	dim3 blocks((int)ceil((float)f[0].cols / (BLOCK_SIZE_16 - 2)), (int)ceil((float) f[0].rows / (BLOCK_SIZE_16 - 2)));
//	
//	hipEvent_t start, stop;
//	hipEventCreate(&start);
//	hipEventCreate(&stop);
//
//	hipEventRecord(start);
//	generate_csrValA_b_kernel<<<blocks, threads>>>(m_d_csrValA, m_d_b, dImg, dPhi0, f[0].cols, f[0].rows);
//	getLastCudaError("generate_csrValA_b_kernel launch failed!");
//	hipEventRecord(stop);
//	hipEventSynchronize(stop);
//
//	float ftime;
//	hipEventElapsedTime(&ftime, start, stop);
//	std::cout << "csvValA_b_kernel running time is: " << ftime << "ms" << std::endl;
//
//	float *h_csrValA = (float*)malloc(sizeof(float)* f[0].cols* f[0].rows * 9);
//
//	checkCudaErrors(hipMemcpy(h_csrValA, m_d_csrValA, sizeof(float)*f[0].cols* f[0].rows * 9, hipMemcpyDeviceToHost));
//
//	std::ofstream out("csrA.csv", std::ios::out | std::ios::trunc);
//
//	for (int i = 0; i < f[0].cols* f[0].rows; i++)
//	{
//		for (int j = 0; j < 3; j++)
//		{
//			for (int k = 0; k < 3; k++)
//			{
//				out << h_csrValA[i * 9 + j * 3 + k]<<",";
//			}
//			out<<"\n";
//		}
//		out<<"\n";
//	}
//
//	out.close();
//
//	free(h_csrValA);
//
//	hipFree(dImg);
//	hipFree(dPhi0);
//}