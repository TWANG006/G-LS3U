#include "hip/hip_runtime.h"
#include "WFT2_CUDAf.h"
#include ""
#include "mem_manager.h"

#include <iostream>
#include <algorithm>
#include <fstream>
#define _USE_MATH_DEFINES
#include <math.h>

namespace WFT_FPA{
namespace WFT{

__inline__ __device__
float warpReduceSum(float val) {
	for (int offset = warpSize / 2; offset > 0; offset /= 2)
		val += __shfl_down(val, offset);
	return val;
}
/*---------------------------------------------CUDA Kernels-------------------------------------------------*/
/* 
 PURPOSE: 
	1. Generate the xf and yf for analytically computation of the Gaussian Window in Fourier Domain 
	[yf xf]=meshgrid(-fix(nn/2):nn-fix(nn/2)-1,-fix(mm/2):mm-fix(mm/2)-1); mm, nn are padded height&width
 NOTE: 
	Currently only even size in each dimension is supported
 INPUTS:
	width, height: width and height of the xf and yf matrices
 OUTPUTS:
	d_out_xf, d_out_yf: Generated meshgrid
  */
__global__ 
void gen_xf_yf_Kernel(hipfftReal *d_out_xf, 
					  hipfftReal *d_out_yf, 
					  int iWidth, int iHeight)
{
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	int j = threadIdx.x + blockIdx.x * blockDim.x;

	int id = i*iWidth + j;

	int iHalfy = iHeight / 2;
	int iHalfx = iWidth / 2;

	if (i < iHeight && j < iWidth)
	{
		d_out_xf[id] = j - iHalfx;
		d_out_yf[id] = i - iHalfy;
	}
}
/*
 PURPOSE:
	2. Do the fftshift on xf and yf to be coincide with the CUFFT's results
 NOTE:
	Currently only even size in each dimension is supported 
 INPUTS:
	width, height: width and height of the xf and yf matrices
 OUTPUTS:
	d_out_xf, d_out_yf: In-place fft-shifted xf, yf

*/
__global__ 
void fftshift_xf_yf_kernel(hipfftReal *d_out_xf, 
						   hipfftReal *d_out_yf,
						   int iWidth, int iHeight)
{
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	int j = threadIdx.x + blockIdx.x * blockDim.x;

	int id = i*iWidth + j;

	int iHalfx = iWidth / 2;
	int iHalfy = iHeight / 2;
	int iSlice = iWidth * iHeight;

	int idQ13 = iSlice / 2 + iHalfx;
	int idQ24 = iSlice / 2 - iHalfx;

	hipfftReal Tempx, Tempy;

	if (j < iHalfx && i < iHalfy)
	{
		Tempx = d_out_xf[id];
		Tempy = d_out_yf[id];

		// First Quadrant
		d_out_xf[id] = d_out_xf[id + idQ13];
		d_out_yf[id] = d_out_yf[id + idQ13];

		// Third Quadrant
		d_out_xf[id + idQ13] = Tempx;
		d_out_yf[id + idQ13] = Tempy;
	}
	else if (j >= iHalfx && j < iWidth && i < iHalfy)
	{

		Tempx = d_out_xf[id];
		Tempy = d_out_yf[id];

		// Second Quadrant
		d_out_xf[id] = d_out_xf[id + idQ24];
		d_out_yf[id] = d_out_yf[id + idQ24];

		// Fourth Quadrant
		d_out_xf[id + idQ24] = Tempx;
		d_out_yf[id + idQ24] = Tempy;		
	}
}
/*
 PURPOSE:
	Feed the input f into the Padded matrix m_d_fPadded 
 INPUTS:
	d_in_f: The input fringe pattern
	iWidth, iHeight: size of the d_f
	iPaddedWidth, iPaddedHeight: FFT preferred size after padding
 OUTPUTS:
	d_out_fPadded: The padded d_f
*/
__global__ 
void feed_fPadded_kernel(hipfftComplex *d_in_f, 
						 hipfftComplex *d_out_fPadded, 
						 int iWidth, int iHeight, 
						 int iPaddedWidth, int iPaddedHeight)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	int idImg = y * iWidth + x;
	int idPadded = y * iPaddedWidth + x;

	if (y < iPaddedHeight && x < iPaddedWidth)
	{
		if (y < iHeight && x < iWidth)
		{
			d_out_fPadded[idPadded].x = d_in_f[idImg].x;
			d_out_fPadded[idPadded].y = d_in_f[idImg].y;
		}
		else
		{
			d_out_fPadded[idPadded].x = 0;
			d_out_fPadded[idPadded].y = 0;
		}
	}
}
/*
 PURPOSE:
	Point-wise multiplication of two matrices of complex numbers
 INPUT:
	d_in_a, d_in_b: Two matrices to be multiplied
	iSize: size of the matrices
 OUTPUT:
	d_out_c: The result after multiplication
*/
__global__
void complex_pointwise_multiplication_kernel(hipfftComplex *d_in_a, 
											 hipfftComplex *d_in_b, 
											 int iSize, 
											 hipfftComplex *d_out_c)
{
	for (int i = threadIdx.x + blockIdx.x*blockDim.x;
		 i < iSize;
		 i += blockDim.x*gridDim.x)
	{
		d_out_c[i] = 
			WFT_FPA::Utils::ComplexScale(WFT_FPA::Utils::ComplexMul(d_in_a[i], d_in_b[i]), 1.0f / iSize);
	}
}
/*
 PURPOSE:
	Explicitly Compute the FFT of the Gaussian Window
		Fg=exp(-(xf*2*pi/mm-wxt).^2/2*sigmax*sigmax - (yf*2*pi/nn-wyt).^2/2*sigmay*sigmay)*sn2;
 INPUTS:
	d_in_xf, d_in_yf: meshgrid in frequency domain
	iPaddedWidth, iPaddedHeight: Padded Gaussian Window size
	wxt, wyt: frequencies in integer intervals
	wxi, wyi: steps
	wxl, wyl: lower bound of the frequencies
	sigmax, sigmay: sigma's in x&y directions
	sn2: normalization params (norm2 = 1);
 OUTPUTS:
	d_out_Fg: Fg
*/
__global__
void compute_Fg_kernel(hipfftReal *d_in_xf, 
					   hipfftReal *d_in_yf, 
					   int iPaddedWidth, int iPaddedHeight, 
					   int wxt, int wyt, float wxi, 
					   float wyi, float wxl, float wyl,
					   float sigmax, float sigmay, 
					   float sn2, 
					   hipfftComplex *d_out_Fg)
{
	hipfftReal rwxt = wxl + hipfftReal(wxt) * wxi;
	hipfftReal rwyt = wyl + hipfftReal(wyt) * wyi;

	for (int i = threadIdx.x + blockIdx.x * blockDim.x;
		 i < iPaddedHeight*iPaddedWidth;
		 i += blockDim.x * gridDim.x)
	{
		hipfftReal tempx = d_in_xf[i] * 2 * hipfftReal(M_PI) * (1.0f / iPaddedWidth) - rwxt;
		hipfftReal tempy = d_in_yf[i] * 2 * hipfftReal(M_PI) * (1.0f / iPaddedHeight) - rwyt;
		tempx = -tempx * tempx * 0.5f * sigmax * sigmax;
		tempy = -tempy * tempy * 0.5f * sigmay * sigmay;
		
		d_out_Fg[i].x = exp(tempx + tempy) * sn2;
		d_out_Fg[i].y = 0;
	}
}

/*-------------------------------------------WFF Specific Utility Kernels-------------------------------------------*/
/*
 PURPOSE:
	Calculate the threshold value for the WFF if it's not specified using Parallel Reduction Algorithm
	thr = 6*sqrt(mean2(abs(f).^2)/3);
 INPUTS:
	d_in:	 type of hipfftComplex input array
	size: size(width*height) of the in
 OUTPUS:
	d_out: 1-element device array
*/
__global__ 
void compute_WFF_threshold_kernel(hipfftComplex *d_in, float *d_out, int size)
{
	float sum = float(0);

	for (int i = threadIdx.x + blockIdx.x * blockDim.x;
		 i < size;
		 i += blockDim.x*gridDim.x)
	{
		float abs = hipCabsf(d_in[i]);
		sum += abs*abs;
	}

	sum=warpReduceSum(sum);

	if (threadIdx.x % warpSize == 0)
		atomicAdd(d_out, sum);
}
/*
 PURPOSE:
	Initialize all WFF related matrices to 0's
 INPUTS:
	iWidth, iHeight: size of the final results
 OUTPUTS:
	d_out_filtered:
*/
__global__ 
void init_WFF_matrices_kernel(hipfftComplex *d_out_filtered, 
							  int iWidth, int iHeight)
{
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int x = threadIdx.x + blockIdx.x * blockDim.x;

	int idImg = y * iWidth + x;

	if (y < iHeight && x < iWidth)
	{
		d_out_filtered[idImg].x = 0;
		d_out_filtered[idImg].y = 0;
	}
}
/*
 PURPOSE:
	Threshold the spectrum sf
 INPUTS:
	iWidth, iHeight: image size
	iPaddedWidth, iPaddedHeight: Padded size
	thr: the threshold
 OUTPUTS:
	d_out_sf: sf after threshold
*/
__global__
void threshold_sf_kernel(hipfftComplex *d_out_sf, 
						 int iWidth, int iHeight, 
						 int iPaddedWidth, int iPaddedHeight, 
						 float thr)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	int idPadded = y * iPaddedWidth + x;

	if (y < iPaddedHeight && x < iPaddedWidth)
	{
		if (y < iHeight && x < iWidth)
		{
			if (hipCabsf(d_out_sf[idPadded]) < thr)
			{
				d_out_sf[idPadded].x = 0;
				d_out_sf[idPadded].y = 0;
			}
		}
		else
		{
			d_out_sf[idPadded].x = 0;
			d_out_sf[idPadded].y = 0;
		}
	}
}
/*
 PURPOSE:
	Update the partial results im_d_filtered of each stream
 INPUTS:
	iWidth, iHeight: image size
	iPaddedWidth, iPaddedHeight: Padded size
	d_in_im_sf: spectrum of each stream
 OUTPUTS:
	d_out_im_filtered: filtered image after of each stream
*/
__global__
void update_WFF_partial_filtered_kernel(hipfftComplex *d_in_im_sf,
										int iWidth, int iHeight, 
										int iPaddedWidth, int iPaddedHeight, 
										hipfftComplex *d_out_im_filtered)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	int idPadded = y * iPaddedWidth + x;
	int idImg = y*iWidth + x;

	if (y < iHeight && x < iWidth)	
	{
		d_out_im_filtered[idImg].x += d_in_im_sf[idPadded].x;
		d_out_im_filtered[idImg].y += d_in_im_sf[idPadded].y;
	}
}
/*
 PURPOSE:
	Update the final z.filtered 
 INPUTS:
	d_in_im_filtered: the partial filtered results in each stream
	imgSize: size of the fringe pattern
 OUTPTS:
	d_out_filtered: the final results
*/
__global__
void update_WFF_final_filtered_kernel(hipfftComplex *d_in_im_filtered, 
									  int imgSize, 
									  hipfftComplex *d_out_filtered)
{
	for (int i = threadIdx.x + blockIdx.x*blockDim.x;
		 i < imgSize;
		 i += blockDim.x * gridDim.x)
	{
		d_out_filtered[i].x += d_in_im_filtered[i].x;
		d_out_filtered[i].y += d_in_im_filtered[i].y;
	}
}
/*
 PURPOSE:
	Scale the final results 
 INPUTS:
	d_out_filtered: the unscaled final results
	imagSize: size of the fringe pattern
	wxi,wyi: step size of the frequencies
 OUTPUT:
	d_out_filtered: scaled final results
*/
__global__
void scale_WFF_final_filtered_kernel(hipfftComplex *d_out_filtered, 
									 int imgSize, 
									 float wxi, float wyi)
{
	float factor = 0.25f * (1.0f / float(M_PI*M_PI)) * wxi * wyi;

	for (int i = threadIdx.x + blockIdx.x*blockDim.x;
		 i < imgSize;
		 i += blockDim.x * gridDim.x)
	{
		d_out_filtered[i].x *= factor;
		d_out_filtered[i].y *= factor;
	}
}
/*----------------------------------------/End WFF Specific Utility Kernels------------------------------------------*/

/*-------------------------------------------WFR Specific Utility Kernels------------------------------------------*/
/*
 PURPOSE:
	Preompute the g used to compute the x.*g, y.*g, cxx&cyy using LS
 INPUTS:
	iWinWidth, iWinHeight: Gaussian Window Size
	iPaddedWidth, iPaddedHeight: Padded size of xg, yg
	sigmax, sigmay: sigma's
 OUTPUTS:
	d_out_g
*/
__global__
void precompute_g_kernel(hipfftReal *d_out_g, 
						 int iWinWidth, int iWinHeight, 
						 float sigmax, float sigmay)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	int iHW = (iWinWidth - 1) / 2;
	int iHH = (iWinHeight - 1) / 2;

	float xx, yy;

	int idWin = y * iWinWidth + x;

	if (y < iWinHeight && x < iWinWidth)
	{
		xx = float(x - iHW);
		yy = float(y - iHH);

		d_out_g[idWin] = exp(-xx*xx*0.5f*(1.0f / (sigmax*sigmax)) - yy*yy*0.5f*(1.0f / (sigmay*sigmay)));
	}
}
/*
 PURPOSE:
	Compute the (sum(sum(g.*g)))
 INPUTS:
	iWinSize: size of the gaussian window
	d_in_g: g
 OUTPUS:
	d_out_norm2g
*/
__global__
void precompute_norm2g_kernel(hipfftReal *d_in_g, int iWinSize, float *d_out_norm2g)
{
	float sum = float(0);

	for (int i = threadIdx.x + blockIdx.x * blockDim.x;
		 i < iWinSize;
		 i += blockDim.x *gridDim.x)
	{
		float tempSqr = d_in_g[i] * d_in_g[i];
		sum += tempSqr;
	}

	sum = warpReduceSum(sum);

	if (threadIdx.x % warpSize == 0)
		atomicAdd(d_out_norm2g, sum);
}
/*
 PURPOSE: 
	compute nomalized g
 INPUTS:
	d_in_norm2g: normalization factor
	iWinszie: Gaussian windows size
 OUTPUTS:
	d_out_g: normalized g
 */
__global__
void precompute_normalized_g_kernel(float *d_in_norm2g, int iWinSize, hipfftReal *d_out_g)
{
	for (int i = threadIdx.x + blockIdx.x * blockDim.x;
		 i < iWinSize;
		 i += blockDim.x * gridDim.x)
	{
		d_out_g[i] = d_out_g[i] * (1.0f / sqrt(d_in_norm2g[0]));
	}
}
/*
 PURPOSE:
	Precompute xg & yg
 INPUTS:
	iWinWidth, iWinHeight: size of the Gaussian Window
	iPaddedWidth, iPaddedHeight: padded size
	d_in_g: the Gaussian Window
 OUTPUS:
	d_out_xg, d_out_yg: the constructed xg&yg
*/
__global__
void precompute_xg_yg_kernel(hipfftReal *d_in_g,
							 int iWinWidth, int iWinHeight, 
							 int iPaddedWidth, int iPaddedHeight, 
							 hipfftComplex *d_out_xg, 
							 hipfftComplex *d_out_yg)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	int idPadded = y * iPaddedWidth + x;
	int idWin = y * iWinWidth + x;

	int iHW = (iWinWidth - 1) / 2;
	int iHH = (iWinHeight - 1) / 2;

	if (y < iPaddedHeight && x < iPaddedWidth)
	{
		if (y < iWinHeight && x < iWinWidth)
		{
			float xx = float(x - iHW);
			float yy = float(y - iHH);

			d_out_xg[idPadded].x = xx * d_in_g[idWin];
			d_out_yg[idPadded].x = yy * d_in_g[idWin];
		}
		else
		{
			d_out_xg[idPadded].x = 0;
			d_out_yg[idPadded].x = 0;
		}
		d_out_xg[idPadded].y = 0;
		d_out_yg[idPadded].y = 0;
	}
}
/*
 PURPOSE:
	Precompute the sum(x.*x.*g) or sum(y.*y.*g)
 INPUTS:
	d_in_xg, d_y_yg: the calculated x.*g, y.*g
	iWinWidth, iWinHeight: Width & Height of the Gaussian Window
	iPaddedWidth, iPaddedHeight: Padded size
 OUTPUS:
	d_out_sumxxg, d_out_sumyyg: sum of x.*x.*g & y.*y.*g	
*/
__global__
void precompute_sum_xxg_yyg_kernel(hipfftComplex *d_in_xg, 
								   hipfftComplex *d_in_yg, 
								   int iWinWidth, int iWinHeight, 
								   int iPaddedWidth, int iPaddedHeight,
								   float *d_out_sumxxg, 
								   float *d_out_sumyyg)
{
	float sumxxg = float(0);
	float sumyyg = float(0);

	for (int i = threadIdx.x + blockIdx.x * blockDim.x;
		 i < iWinHeight * iWinWidth;
		 i += gridDim.x * blockDim.x)
	{
		int x = i % iWinWidth;
		int y = i / iWinWidth;

		float xx = float(x - iWinWidth);
		float yy = float(y - iWinHeight);
		
		float tempxg = d_in_xg[y*iPaddedWidth + x].x * xx;
		float tempyg = d_in_yg[y*iPaddedWidth + x].x * yy;

		sumxxg += tempxg;
		sumyyg += tempyg;
	}

	sumxxg = warpReduceSum(sumxxg);
	sumyyg = warpReduceSum(sumyyg);

	if (threadIdx.x % warpSize == 0)
	{
		atomicAdd(d_out_sumxxg, sumxxg);
		atomicAdd(d_out_sumyyg, sumyyg);
	}	
}
/*
 PURPOSE: 
	Initialize the final results of WFR to zero's
 INPUTS:
	imgSize: the image sizes
 OUTPUTS:
	d_out_wx, d_out_wy, d_out_phase, d_out_phase_comp, d_out_r, d_out_b, d_out_cxx, d_out_cyy: to be initialized 
*/
__global__
void initialize_WFR_final_results_kernel(int iImgSize,
										 hipfftReal* d_out_wx, 
										 hipfftReal* d_out_wy,
										 hipfftReal* d_out_phase,
										 hipfftReal* d_out_phase_comp,
										 hipfftReal* d_out_r,
										 hipfftReal* d_out_b, 
										 hipfftReal* d_out_cxx,
										 hipfftReal* d_out_cyy)
{
	for (int i = threadIdx.x + blockIdx.x*blockDim.x;
		 i < iImgSize;
		 i += gridDim.x * blockDim.x)
	{
		d_out_wx[i] = 0;
		d_out_wy[i] = 0;
		d_out_phase[i] = 0;
		d_out_phase_comp[i] = 0;
		d_out_cxx[i] = 0;
		d_out_cyy[i] = 0;
		d_out_b[i] = 0;
		d_out_r[i] = 0;
	}
}
/*
 PURPOSE:
	Initialize per-stream intermediate results
 INPUTS:
	iPaddedSize: padded size 
 OUTPUS:
	d_out_im_wx, d_out_im_wy, d_out_im_p, d_out_im_r: to be initialized
*/
__global__
void initialize_WFR_im_results_kernel(int iImgSize,
									  hipfftReal* d_out_im_wx, 
									  hipfftReal* d_out_im_wy, 
									  hipfftReal* d_out_im_p, 
									  hipfftReal* d_out_im_r)
{
	for (int i = threadIdx.x + blockIdx.x*blockDim.x;
		 i < iImgSize;
		 i += gridDim.x * blockDim.x)
	{
		d_out_im_wx[i] = 0;
		d_out_im_wy[i] = 0;
		d_out_im_p[i] = 0;
		d_out_im_r[i] = 0;
	}
}
/*
 PURPOSE:
	Update the r, wx, wy and p of each iteration
 INPUTS:
	d_in_sf: the computed sf
	wxl, wyl: lower-bound of the frequencies
	wxt, wyt: current frequency
	wxi, wyi: step size of the frequencies
	iPaddedSize: padded size
 OUTPUTS:
	d_out_r, d_out_wx, d_out_wy, d_out_p: updated
*/
__global__
void update_r_wx_wy_p_kernel(hipfftComplex *d_in_sf,
							 int wxt, float wxl, float wxi, 
							 int wyt, float wyl, float wyi, 
							 int iPaddedWidth, int iPaddedHeight, 
							 int iImgWidth, int iImgHeight,
							 hipfftReal* d_out_r, 
							 hipfftReal* d_out_wx, 
							 hipfftReal* d_out_wy, 
							 hipfftReal* d_out_p)
{
	float rwxt = wxl + float(wxt) * wxi;
	float rwyt = wyl + float(wyt) * wyi;

	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	int idImg = y * iImgWidth + x;
	int idPadded = y * iPaddedWidth + x;

	if (y < iImgHeight && x < iImgWidth)
	{
		float abs_sf = hipCabsf(d_in_sf[idPadded]);
		
		if (abs_sf > d_out_r[idImg])
		{
			d_out_r[idImg] = abs_sf;
			d_out_wx[idImg] = rwxt;
			d_out_wy[idImg] = rwyt;
			d_out_p[idImg] = atan2f(d_in_sf[idPadded].y, d_in_sf[idPadded].x);
		}
	}
}
/*
 PURPOSE: 
	Update the final r, wx, wy and p 
 INPUTS:
	d_in_r, d_in_wx, d_in_wy, d_in_p: partial results of each stream
	iImgSize: image size
 OUTPUTS:
	d_out_r, d_out_wx, d_out_wy, d_out_p: final results
*/
__global__
void update_final_r_wx_wy_p_kernel(hipfftReal* d_in_r, 
								   hipfftReal* d_in_wx, 
								   hipfftReal* d_in_wy, 
								   hipfftReal* d_in_p,
								   int iImgSize,
								   hipfftReal* d_out_r, 
								   hipfftReal* d_out_wx, 
								   hipfftReal* d_out_wy, 
								   hipfftReal* d_out_p)
{
	for (int i = threadIdx.x + blockDim.x * blockIdx.x;
		 i < iImgSize;
		 i += gridDim.x * blockDim.x)
	{
		if(d_in_r[i] > d_out_r[i])
		{
			d_out_r[i] = d_in_r[i];
			d_out_wx[i] = d_in_wx[i];
			d_out_wy[i] = d_in_wy[i];
			d_out_p[i] = d_in_p[i];
		}
	}
}
/*
 PURPOSE:
	Feed the wx&wy into padded cxx&cyy
 INPUTS:
	d_in_wx, d_in_wy
	iWidth, iHeight: image size
	iPaddedWidth, iPaddedHeight: padded size
 OUTPUTS:
	d_out_cxx, d_out_cyy: padded cxx&cyy
*/
__global__
void feed_cxx_cyy_kernel(hipfftReal* d_in_wx,
						 hipfftReal* d_in_wy, 
						 int iWidth, int iHeight, 
						 int iPaddedWidth, int iPaddedHeight,
						 hipfftComplex *d_out_cxx, 
						 hipfftComplex *d_out_cyy)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	int idPadded = y * iPaddedWidth + x;
	int idImg = y* iWidth + x;

	if (y < iPaddedHeight && x < iPaddedWidth)
	{
		if (y < iHeight && x < iWidth)
		{
			d_out_cxx[idPadded].x = d_in_wx[idImg];
			d_out_cyy[idPadded].x = d_in_wy[idImg];
		}
		else
		{
			d_out_cxx[idPadded].x = 0;
			d_out_cyy[idPadded].x = 0;
		}
		d_out_cxx[idPadded].y = 0;
		d_out_cyy[idPadded].y = 0;
	}
}
/*
 PURPOSE:
	2D Point-wise multiplication of two matrices of complex numbers
 INPUT:
	d_in_a1, d_in_b1, d_in_a2, d_in_b2: Two sets of matrices to be multiplied
	iSize: size of the matrices
 OUTPUT:
	d_out_c1, d_out_c2: The results after multiplication
*/
__global__
void complex_pointwise_multiplication_2d_kernel(hipfftComplex *d_in_a1, 
												hipfftComplex *d_in_b1, 
												hipfftComplex *d_in_a2, 
												hipfftComplex *d_in_b2,
												int iSize, 
												hipfftComplex *d_out_c1, 
												hipfftComplex *d_out_c2)
{
	for (int i = threadIdx.x + blockIdx.x*blockDim.x;
		 i < iSize;
		 i += blockDim.x*gridDim.x)
	{
		d_out_c1[i] = 
			WFT_FPA::Utils::ComplexScale(WFT_FPA::Utils::ComplexMul(d_in_a1[i], d_in_b1[i]), 1.0f / iSize);
		d_out_c2[i] = 
			WFT_FPA::Utils::ComplexScale(WFT_FPA::Utils::ComplexMul(d_in_a2[i], d_in_b2[i]), 1.0f / iSize);
	}
}
/*
 PURPOSE: 
	Update the results after compensation
 INPUTS:
	d_in_wx, d_in_wy, d_in_r, d_in_p: calculated results used to update the compensated phase
 OUTPUTS:
	d_out_cxx, d_out_cyy, d_out_phase_comp, d_out_b: results 
*/
__global__
void update_final_cxx_cyy_phaseComp_b_kernel(hipfftComplex* d_in_cxx,
											 hipfftComplex* d_in_cyy, 
											 hipfftReal* d_in_r, 
											 hipfftReal* d_in_p,
											 int iWidth, int iHeight,
											 int iPaddedWidth, int iPaddedHeight, 
											 float sumxxg, float sumyyg, 
											 float sigmax, float sigmay, 
											 int sx, int sy,
											 hipfftReal* d_out_cxx, 
											 hipfftReal* d_out_cyy, 
											 hipfftReal* d_out_phase_comp, 
											 hipfftReal* d_out_b)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	int idImg = y * iWidth + x;
	int idPadded = (y + sy) * iPaddedWidth + x + sx;

	if (x < iWidth && y < iHeight)
	{
		hipfftReal cxx = d_in_cxx[idPadded].x;
		hipfftReal cyy = d_in_cyy[idPadded].x;

		// curvature estimation
		hipfftReal tempCxx = -cxx * (1.0f / sumxxg);
		hipfftReal tempCyy = -cyy * (1.0f / sumyyg);

		d_out_cxx[idImg] = tempCxx;
		d_out_cyy[idImg] = tempCyy;
		
		// phase compensation
		hipfftReal tempPhaseComp =  d_in_p[idImg] - 0.5f * atanf(sigmax*sigmax*tempCxx) - 0.5f * atanf(sigmay*sigmay*tempCyy);
		d_out_phase_comp[idImg] = atan2f(sin(tempPhaseComp), cos(tempPhaseComp));

		//scale amplitude
		d_out_b[idImg] = d_in_r[idImg] 
			* powf((1 + sigmax*sigmax*sigmax*sigmax*cxx*cxx)*0.25f*(1.0f / M_PI)*(1.0f / (sigmax*sigmax)), 0.25f)
			* powf((1 + sigmay*sigmay*sigmay*sigmay*cyy*cyy)*0.25f*(1.0f / M_PI)*(1.0f / (sigmay*sigmay)), 0.25f);
	}
}

/*----------------------------------------/End WFR Specific Utility Kernels------------------------------------------*/


/*------------------------------------------------/End CUDA Kernels--------------------------------------------------*/




/*--------------------------------------------------WFT2 Implementations-----------------------------------------------*/
WFT2_CUDAF::WFT2_CUDAF(int iWidth, int iHeight,
					   WFT_TYPE type,
					   WFT2_DeviceResultsF& z,
					   int iNumStreams)
	: m_iWidth(iWidth)
	, m_iHeight(iHeight)
	, m_type(type)
	, m_rThr(-1)
	, m_d_rThr(nullptr)
	, m_iNumStreams(iNumStreams)
	, m_cudaStreams(nullptr)
	, m_d_fPadded(nullptr)
	, m_d_xf(nullptr)
	, m_d_yf(nullptr)
	, im_d_Fg(nullptr)
	, im_d_filtered(nullptr)
	, im_d_r(nullptr)
	, im_d_p(nullptr)
	, im_d_wx(nullptr)
	, im_d_wy(nullptr)
	, im_d_cxxPadded(nullptr)
	, im_d_cyyPadded(nullptr)
	, im_d_xgPadded(nullptr)
	, im_d_ygPadded(nullptr)
	, m_planStreams(nullptr)
{
	// Check the input image size
	//if (iWidth % 2 != 0 || iHeight % 2 != 0)
	//{
	//	std::cout << "GPU implementation of WFT curretly only suppports even image size!" << std::endl;
	//	throw -2;

	//}
	
	// Get the number of SMs on GPU 
	hipDeviceGetAttribute(&m_iSMs, hipDeviceAttributeMultiprocessorCount, 0);

	/* Type specific parameter initializations*/
	if (WFT_TYPE::WFF == m_type)
	{
		m_rSigmaX = 10.0f;
		m_rWxl = -2.0f - 3.0f / m_rSigmaX;
		m_rWxi = 1.0f / m_rSigmaX;
		m_rWxh = 2.0f + 3.0f / m_rSigmaX;

		m_rSigmaY = 10.0f;
		m_rWyl = -2.0f - 3.0f / m_rSigmaY;
		m_rWyi = 1.0f / m_rSigmaY;
		m_rWyh = 2.0f + 3.0f / m_rSigmaY;
	}
	else if (WFT_TYPE::WFR == m_type)
	{
		m_rSigmaX = 10.0f;
		m_rWxl = -2.0f;
		m_rWxi = 0.025f;
		m_rWxh = 2.0f;

		m_rSigmaY = 10.0f;
		m_rWyl = -2.0f;
		m_rWyi = 0.025f;
		m_rWyh = 2.0f;
	}

	// scale for window so that norm2 of the window is 1. 
	m_rGaussianNorm2 = sqrt(4 * float(M_PI)*m_rSigmaX*m_rSigmaY);

	/* Do the Initialization */
	if(-1 == cuWFT2_Initialize(z))
	{
		std::cout<<"FFT padding is out of range [4096]. Shrink the size of either the image or the Gaussian Window!"<<std::endl;
		throw -1;
	}
}

WFT2_CUDAF::WFT2_CUDAF(int iWidth, int iHeight,
					   WFT_TYPE type,
					   float rSigmaX, float rWxl, float rWxh, float rWxi,
					   float rSigmaY, float rWyl, float rWyh, float rWyi,
					   float rThr,
					   WFT2_DeviceResultsF &z,
					   int iNumStreams)
	: m_iWidth(iWidth)
	, m_iHeight(iHeight)
	, m_type(type)
	, m_rSigmaX(rSigmaX)
	, m_rSigmaY(rSigmaY)
	, m_rWxl(rWxl)
	, m_rWxi(rWxi)
	, m_rWxh(rWxh)
	, m_rWyl(rWxl)
	, m_rWyi(rWyi)
	, m_rWyh(rWyh)
	, m_rThr(rThr)
	, m_d_rThr(nullptr)
	, m_iNumStreams(iNumStreams)
	, m_cudaStreams(nullptr)
	, m_d_fPadded(nullptr)
	, m_d_xf(nullptr)
	, m_d_yf(nullptr)
	, im_d_Fg(nullptr)
	, im_d_filtered(nullptr)
	, im_d_r(nullptr)
	, im_d_p(nullptr)
	, im_d_wx(nullptr)
	, im_d_wy(nullptr)
	, im_d_cxxPadded(nullptr)
	, im_d_cyyPadded(nullptr)
	, im_d_xgPadded(nullptr)
	, im_d_ygPadded(nullptr)
	, m_planStreams(nullptr)
{
	// Check the input image size
	//if (iWidth % 2 != 0 || iHeight % 2 != 0)
	//{
	//	std::cout << "GPU implementation of WFT curretly only suppports even image size!" << std::endl;
	//	throw -2;

	//}
	
	// Get the number of SMs on GPU 
	hipDeviceGetAttribute(&m_iSMs, hipDeviceAttributeMultiprocessorCount, 0);

	// scale for window so that norm2 of the window is 1. 
	m_rGaussianNorm2 = sqrt(4 * float(M_PI)*m_rSigmaX*m_rSigmaY);
	/* Do the Initialization */
	if (-1 == cuWFT2_Initialize(z))
	{
		std::cout << "FFT padding is out of range [4096]. Shrink the size of either the image or the Gaussian Window!" << std::endl;
		throw - 1;
	}
}

WFT2_CUDAF::~WFT2_CUDAF()
{
	WFT_FPA::Utils::cudaSafeFree(m_d_fPadded);
	WFT_FPA::Utils::cudaSafeFree(m_d_xf);
	WFT_FPA::Utils::cudaSafeFree(m_d_yf);

	hipfftDestroy(m_planPadded);

	if (WFT_FPA::WFT::WFT_TYPE::WFF == m_type)
	{
		// Destroy stream-specific stuffs
		for (int i = 0; i < m_iNumStreams; i++)
		{
			hipStreamDestroy(m_cudaStreams[i]);
			hipfftDestroy(m_planStreams[i]);
			WFT_FPA::Utils::cudaSafeFree(im_d_filtered[i]);
			WFT_FPA::Utils::cudaSafeFree(im_d_Fg[i]);
			WFT_FPA::Utils::cudaSafeFree(im_d_Sf[i]);
		}
		free(m_cudaStreams);			m_cudaStreams = nullptr;
		free(m_planStreams);			m_planStreams = nullptr;
		free(im_d_filtered);			im_d_filtered = nullptr;
		free(im_d_Fg);					im_d_Fg = nullptr;
		free(im_d_Sf);					im_d_Sf = nullptr;

		WFT_FPA::Utils::cudaSafeFree(m_d_rThr);
	}

	if (WFT_FPA::WFT::WFT_TYPE::WFR == m_type)
	{
		for (int i = 0; i < m_iNumStreams; i ++)
		{
			hipStreamDestroy(m_cudaStreams[i]);
			hipfftDestroy(m_planStreams[i]);
			WFT_FPA::Utils::cudaSafeFree(im_d_wx[i]);
			WFT_FPA::Utils::cudaSafeFree(im_d_wy[i]);
			WFT_FPA::Utils::cudaSafeFree(im_d_p[i]);
			WFT_FPA::Utils::cudaSafeFree(im_d_r[i]);
		}

		free(m_cudaStreams);	m_cudaStreams = nullptr;
		free(m_planStreams);	m_planStreams = nullptr;
		free(im_d_wx);	im_d_wx = nullptr;
		free(im_d_wy);	im_d_wy = nullptr;
		free(im_d_p);	im_d_p = nullptr;
		free(im_d_r);	im_d_r = nullptr;

		WFT_FPA::Utils::cudaSafeFree(im_d_cxxPadded);
		WFT_FPA::Utils::cudaSafeFree(im_d_cyyPadded);
		WFT_FPA::Utils::cudaSafeFree(im_d_xgPadded);
		WFT_FPA::Utils::cudaSafeFree(im_d_ygPadded);
		WFT_FPA::Utils::cudaSafeFree(m_d_rg_norm2);
		WFT_FPA::Utils::cudaSafeFree(m_d_rxxg_norm2);
		WFT_FPA::Utils::cudaSafeFree(m_d_ryyg_norm2);
	}
}

void WFT2_CUDAF::operator()(hipfftComplex *d_f,
							WFT2_DeviceResultsF &d_z,
							double &time)
{
	if (WFT_FPA::WFT::WFT_TYPE::WFF == m_type)
		cuWFF2(d_f, d_z, time);
	else if (WFT_FPA::WFT::WFT_TYPE::WFR == m_type)
		cuWFR2(d_f, d_z, time);
}


/* Private functions */

void WFT2_CUDAF::cuWFF2(hipfftComplex *d_f, WFT2_DeviceResultsF &d_z, double &time)
{
	/* CUDA blocks & threads scheduling */
	dim3 threads(BLOCK_SIZE_16, BLOCK_SIZE_16);
	dim3 blocksPadded((m_iPaddedWidth + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16, (m_iPaddedHeight + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16);
	dim3 blocksImg((m_iWidth + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16, (m_iHeight + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16);
	int blocks1D = std::min((m_iPaddedWidth*m_iPaddedHeight + BLOCK_SIZE_256 - 1) / BLOCK_SIZE_256, 2048);

	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	/* Set the threshold m_rThr if it's not specified by the client */

	hipEventRecord(start);
	cuWFF2_SetThreashold(d_f);

	/* Feed the f to its padded version */
	cuWFT2_feed_fPadded(d_f);
	
	/* Pre-compute the FFT of m_d_fPadded */
	checkCudaErrors(hipfftExecC2C(m_planPadded, m_d_fPadded, m_d_fPadded, HIPFFT_FORWARD));

	/*hipfftComplex *h_fPadded = (hipfftComplex*)malloc(sizeof(hipfftComplex) * m_iPaddedWidth * m_iPaddedHeight);
	checkCudaErrors(hipMemcpy(h_fPadded, m_d_fPadded, sizeof(hipfftComplex) * m_iPaddedWidth * m_iPaddedHeight, hipMemcpyDeviceToHost));
	std::ofstream out("device_FfPadded.csv", std::ios::out | std::ios::trunc);

	for (int i = 0; i < m_iPaddedHeight; i++)
	{
		for (int j = 0; j < m_iPaddedWidth; j++)
		{
			out << h_fPadded[i * m_iPaddedWidth + j].x << "+" << h_fPadded[i * m_iPaddedWidth + j].y << "i" << ",";
		}
		out << "\n";
	}
	out.close();

	free(h_fPadded);*/


	/* Clear the results if they already contain last results */	
	init_WFF_matrices_kernel<<<blocksImg, threads>>>(d_z.m_d_filtered, m_iWidth, m_iHeight);
	getLastCudaError("init_WFF_matrices_kernel Launch Failed!");

	/* Insert this part inbetween to realize kind of CPU&GPU concurrent execution.
	   map the wl: wi : wh interval to integers from  0 to size = (wyh - wyl)/wyi + 1 in order to divide the 
	   copmutations across threads, since threads indices are more conviniently controlled by integers 	    */
	int iwx = int((m_rWxh - m_rWxl)*(1 / m_rWxi)) + 1;
	int iwy = int((m_rWyh - m_rWyl)*(1 / m_rWyi)) + 1;

	for (int i = 0; i < m_iNumStreams; i++)
	{
		init_WFF_matrices_kernel<<<blocksPadded, threads, 0, m_cudaStreams[i]>>>(im_d_filtered[i], m_iPaddedWidth, m_iPaddedHeight);
		getLastCudaError("init_WFF_matrices_kernel Launch Failed!");
	}

	/*std::vector<std::thread> td(m_iNumStreams);

	for (int i = 0; i < m_iNumStreams; i++)
	{
		td[i] = (std::thread(init_WFF_matrices, im_d_filtered[i], m_iPaddedWidth, m_iPaddedHeight));
	}
	std::for_each(td.begin(), td.end(), std::mem_fn(&std::thread::join));
	getLastCudaError("init_WFF_matrices_kernel Launch Failed!");*/

	/* Start the Real WFF iterations */
	

	int iNumResidue = iwx % m_iNumStreams;
	
	for (int y = 0; y < iwy; y++)
	{
		// Now we have equal number of kernels executed in each stream
		for (int x = iNumResidue; x < iwx; x += m_iNumStreams)
		{
			for (int i = 0; i < m_iNumStreams; i++)
			{
				// Construct Fg
				compute_Fg_kernel<<<blocks1D, BLOCK_SIZE_256, 0, m_cudaStreams[i]>>>(
					m_d_xf, m_d_yf, m_iPaddedWidth, m_iPaddedHeight,
					x + i, y, m_rWxi, m_rWyi, m_rWxl, m_rWyl,
					m_rSigmaX, m_rSigmaY, m_rGaussianNorm2, im_d_Fg[i]);
				getLastCudaError("compute_Fg_kernel Launch Failed!");
				
				// Compute sf=ifft2(Ff.*Fg)
				complex_pointwise_multiplication_kernel<<<blocks1D, BLOCK_SIZE_256, 0, m_cudaStreams[i]>>>(
					m_d_fPadded, im_d_Fg[i], m_iPaddedHeight*m_iPaddedWidth, im_d_Sf[i]);
				getLastCudaError("complex_pointwise_multiplication_kernel Launch Failed!");
				checkCudaErrors(hipfftExecC2C(m_planStreams[i], im_d_Sf[i], im_d_Sf[i], HIPFFT_BACKWARD));

				// Threshold the sf: sf=sf.*(abs(sf)>=thr); 
				threshold_sf_kernel<<<blocksPadded, threads, 0, m_cudaStreams[i]>>>(im_d_Sf[i], m_iWidth, m_iHeight, m_iPaddedWidth, m_iPaddedHeight, m_rThr);
				getLastCudaError("threshold_sf_kernel Launch Failed!");

				// implement of IWFT: conv2(sf,w);
				checkCudaErrors(hipfftExecC2C(m_planStreams[i], im_d_Sf[i], im_d_Sf[i], HIPFFT_FORWARD));
				complex_pointwise_multiplication_kernel<<<blocks1D, BLOCK_SIZE_256, 0, m_cudaStreams[i]>>>(
					im_d_Sf[i], im_d_Fg[i], m_iPaddedHeight*m_iPaddedWidth, im_d_Sf[i]);
				getLastCudaError("complex_pointwise_multiplication_kernel Launch Failed!");
				checkCudaErrors(hipfftExecC2C(m_planStreams[i], im_d_Sf[i], im_d_Sf[i], HIPFFT_BACKWARD));

				// Update partial results im_d_filtered
				update_WFF_partial_filtered_kernel<<<blocksImg, threads, 0, m_cudaStreams[i]>>>(im_d_Sf[i], m_iWidth, m_iHeight, m_iPaddedWidth, m_iPaddedHeight, im_d_filtered[i]);
				getLastCudaError("update_WFF_partial_filtered_kernel Launch Failed!");
			}
		}
		// Deal with the residues
		for (int x = 0; x < iNumResidue; x++)
		{
			// Construct Fg
			compute_Fg_kernel<<<blocks1D, BLOCK_SIZE_256, 0, m_cudaStreams[x] >>>(
				m_d_xf, m_d_yf, m_iPaddedWidth, m_iPaddedHeight,
				x, y, m_rWxi, m_rWyi, m_rWxl, m_rWyl,
				m_rSigmaX, m_rSigmaY, m_rGaussianNorm2, im_d_Fg[x]);
			getLastCudaError("compute_Fg_kernel Launch Failed!");
			
			// Compute sf=ifft2(Ff.*Fg)
			complex_pointwise_multiplication_kernel<<<blocks1D, BLOCK_SIZE_256, 0, m_cudaStreams[x] >>>(
				m_d_fPadded, im_d_Fg[x], m_iPaddedHeight*m_iPaddedWidth, im_d_Sf[x]);
			getLastCudaError("complex_pointwise_multiplication_kernel Launch Failed!");
			checkCudaErrors(hipfftExecC2C(m_planStreams[x], im_d_Sf[x], im_d_Sf[x], HIPFFT_BACKWARD));

			// Threshold the sf: sf=sf.*(abs(sf)>=thr); 
			threshold_sf_kernel<<<blocksPadded, threads, 0, m_cudaStreams[x]>>>(im_d_Sf[x], m_iWidth, m_iHeight, m_iPaddedWidth, m_iPaddedHeight, m_rThr);
			getLastCudaError("threshold_sf_kernel Launch Failed!");

			// implement of IWFT: conv2(sf,w);
			checkCudaErrors(hipfftExecC2C(m_planStreams[x], im_d_Sf[x], im_d_Sf[x], HIPFFT_FORWARD));
			complex_pointwise_multiplication_kernel<<<blocks1D, BLOCK_SIZE_256, 0, m_cudaStreams[x]>>>(
				im_d_Sf[x], im_d_Fg[x], m_iPaddedHeight*m_iPaddedWidth, im_d_Sf[x]);
			getLastCudaError("complex_pointwise_multiplication_kernel Launch Failed!");
			checkCudaErrors(hipfftExecC2C(m_planStreams[x], im_d_Sf[x], im_d_Sf[x], HIPFFT_BACKWARD));

			// Update partial results im_d_filtered
			update_WFF_partial_filtered_kernel<<<blocksImg, threads, 0, m_cudaStreams[x]>>>(im_d_Sf[x], m_iWidth, m_iHeight, m_iPaddedWidth, m_iPaddedHeight, im_d_filtered[x]);
			getLastCudaError("update_WFF_partial_filtered_kernel Launch Failed!");
		}
	}
	for (int i = 0; i < m_iNumStreams; i++)
	{
		hipStreamSynchronize(m_cudaStreams[i]);
	}

	for (int i = 0; i < m_iNumStreams; i++)
	{
		update_WFF_final_filtered_kernel<<<blocks1D, BLOCK_SIZE_256>>>(im_d_filtered[i], m_iWidth*m_iHeight, d_z.m_d_filtered);
		getLastCudaError("update_WFF_final_filtered_kernel Launch Failed!");
	}
	scale_WFF_final_filtered_kernel<<<blocks1D, BLOCK_SIZE_256>>>(d_z.m_d_filtered, m_iWidth*m_iHeight, m_rWxi, m_rWyi);
	getLastCudaError("scale_WFF_final_filtered_kernel Launch Failed!");

	hipEventRecord(end);
	hipEventSynchronize(end);

	// Calculate the running time
	float t = 0;
	hipEventElapsedTime(&t, start, end);
	time = double(t);
}
void WFT2_CUDAF::cuWFR2(hipfftComplex *d_f, WFT2_DeviceResultsF &d_z, double &time)
{
	/* Various Sizes */
	int iPaddedSize = m_iPaddedHeight * m_iPaddedWidth;
	int iWinSize = m_iWinHeight * m_iWinWidth;
	int iImgSize = m_iWidth * m_iHeight;

	/* CUDA blocks & threads scheduling */
	dim3 threads(BLOCK_SIZE_16, BLOCK_SIZE_16);
	dim3 blocksPadded((m_iPaddedWidth + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16, (m_iPaddedHeight + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16);
	dim3 blocksImg((m_iWidth + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16, (m_iHeight + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16);
	int blocks1D_pad = std::min((iPaddedSize+ BLOCK_SIZE_256 - 1) / BLOCK_SIZE_256, 2048);
	int blocks1D_img = std::min((iImgSize + BLOCK_SIZE_256 - 1) / BLOCK_SIZE_256, 2048);

	/* Pad the f to be prefered size of the FFT */
	cuWFT2_feed_fPadded(d_f);

	/* Pre-compute the FFT of m_d_fPadded */
	hipfftExecC2C(m_planPadded, m_d_fPadded, m_d_fPadded, HIPFFT_FORWARD);

	/* Clear the results if they already contain last results */	
	initialize_WFR_final_results_kernel<<<blocks1D_img, BLOCK_SIZE_256>>>(
		iImgSize, 
		d_z.m_d_wx, d_z.m_d_wy, d_z.m_d_phase, d_z.m_d_phase_comp, d_z.m_d_r, d_z.m_d_b, d_z.m_d_cxx, d_z.m_d_cyy);
	getLastCudaError("initialize_WFR_final_results_kernel Launch Failed!");

	/* Insert this part inbetween to realize kind of CPU&GPU concurrent execution.
	   map the wl: wi : wh interval to integers from  0 to size = (wyh - wyl)/wyi + 1 in order to divide the 
	   copmutations across threads, since threads indices are more conviniently controlled by integers 	    */
	int iwx = int((m_rWxh - m_rWxl)*(1 / m_rWxi)) + 1;
	int iwy = int((m_rWyh - m_rWyl)*(1 / m_rWyi)) + 1;

	for (int i = 0; i < m_iNumStreams; i++)
	{
		initialize_WFR_im_results_kernel<<<blocks1D_img, BLOCK_SIZE_256, 0, m_cudaStreams[i]>>>(
			iImgSize,
			im_d_wx[i], im_d_wy[i], im_d_p[i], im_d_r[i]);
		getLastCudaError("initialize_WFR_im_results_kernel Launch Failed!");
	}

	/* Start the Real WFF iterations */
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);

	int iNumResidue = iwx % m_iNumStreams;
	hipEventRecord(start);
	for (int y = 0; y < iwy; y++)
	{
		// Now we have equal number of kernels executed in each stream
		for (int x = iNumResidue; x < iwx; x += m_iNumStreams)
		{
			for (int i = 0; i < m_iNumStreams; i++)
			{
				// Construct Fg
				compute_Fg_kernel<<<blocks1D_pad, BLOCK_SIZE_256, 0, m_cudaStreams[i]>>>(
					m_d_xf, m_d_yf, m_iPaddedWidth, m_iPaddedHeight,
					x + i, y, m_rWxi, m_rWyi, m_rWxl, m_rWyl,
					m_rSigmaX, m_rSigmaY, m_rGaussianNorm2, im_d_Fg[i]);
				getLastCudaError("compute_Fg_kernel Launch Failed!");
				
				// Compute sf=ifft2(Ff.*Fg)
				complex_pointwise_multiplication_kernel<<<blocks1D_pad, BLOCK_SIZE_256, 0, m_cudaStreams[i]>>>(
					m_d_fPadded, im_d_Fg[i], iPaddedSize, im_d_Sf[i]);
				getLastCudaError("complex_pointwise_multiplication_kernel Launch Failed!");
				checkCudaErrors(hipfftExecC2C(m_planStreams[i], im_d_Sf[i], im_d_Sf[i], HIPFFT_BACKWARD));

				// Update r, wx, wy and phase
				update_r_wx_wy_p_kernel<<<blocksImg, threads, 0, m_cudaStreams[i]>>>(
					im_d_Sf[i], x + i, m_rWxl, m_rWxi, y, m_rWyl, m_rWyi, 
					m_iPaddedWidth, m_iPaddedHeight, m_iWidth, m_iHeight, 
					im_d_r[i], im_d_wx[i], im_d_wy[i], im_d_p[i]);
				getLastCudaError("update_r_wx_wy_p_kernel Launch Failed!");
			}
		}

		for (int x = 0; x < iNumResidue; x++)
		{
			// Construct Fg
			compute_Fg_kernel<<<blocks1D_pad, BLOCK_SIZE_256, 0, m_cudaStreams[x]>>>(
				m_d_xf, m_d_yf, m_iPaddedWidth, m_iPaddedHeight,
				x, y, m_rWxi, m_rWyi, m_rWxl, m_rWyl,
				m_rSigmaX, m_rSigmaY, m_rGaussianNorm2, im_d_Fg[x]);
			getLastCudaError("compute_Fg_kernel Launch Failed!");
			
			// Compute sf=ifft2(Ff.*Fg)
			complex_pointwise_multiplication_kernel<<<blocks1D_pad, BLOCK_SIZE_256, 0, m_cudaStreams[x]>>>(
				m_d_fPadded, im_d_Fg[x], iPaddedSize, im_d_Sf[x]);
			getLastCudaError("complex_pointwise_multiplication_kernel Launch Failed!");
			checkCudaErrors(hipfftExecC2C(m_planStreams[x], im_d_Sf[x], im_d_Sf[x], HIPFFT_BACKWARD));

			// Update r, wx, wy and phase
			update_r_wx_wy_p_kernel<<<blocksImg, threads, 0, m_cudaStreams[x]>>>(
				im_d_Sf[x], x, m_rWxl, m_rWxi, y, m_rWyl, m_rWyi, 
				m_iPaddedWidth, m_iPaddedHeight, m_iWidth, m_iHeight, 
				im_d_r[x], im_d_wx[x], im_d_wy[x], im_d_p[x]);
			getLastCudaError("update_r_wx_wy_p_kernel Launch Failed!");
		}
	}
	// Synchronize streams
	for (int i = 0; i < m_iNumStreams; i++)
	{
		hipStreamSynchronize(m_cudaStreams[i]);
	}

	for (int i = 0; i < m_iNumStreams; i++)
	{
		update_final_r_wx_wy_p_kernel<<<blocks1D_img, BLOCK_SIZE_256>>>(im_d_r[i], im_d_wx[i], im_d_wy[i], im_d_p[i], iImgSize,
			d_z.m_d_r, d_z.m_d_wx, d_z.m_d_wy, d_z.m_d_phase);
		getLastCudaError("update_final_r_wx_wy_p_kernel Launch Failed!");
	}

	hipEventRecord(end);
	hipEventSynchronize(end);

	// Calculate the running time
	float t = 0;
	hipEventElapsedTime(&t, start, end);
	time = double(t);

	/* Do the Least squre fitting to get cx and cy */
	/* Feed the wx & wy into their padded versions*/
	feed_cxx_cyy_kernel<<<blocksPadded, threads>>>(d_z.m_d_wx, d_z.m_d_wy, m_iWidth, m_iHeight, m_iPaddedWidth, m_iPaddedHeight,
		im_d_cxxPadded, im_d_cyyPadded);
	getLastCudaError("feed_cxx_cyy_kernel Launch Failed!");

	// z.cxx=-conv2(z.wx,x.*g,'same')/sum(sum(x.*x.*g));
    // z.cyy=-conv2(z.wy,y.*g,'same')/sum(sum(y.*y.*g)); 
	// Forward FFT
	checkCudaErrors(hipfftExecC2C(m_planPadded, im_d_cxxPadded, im_d_cxxPadded, HIPFFT_FORWARD));
	checkCudaErrors(hipfftExecC2C(m_planPadded, im_d_cyyPadded, im_d_cyyPadded, HIPFFT_FORWARD));
	// Pointwise multiplication
	complex_pointwise_multiplication_2d_kernel<<<blocks1D_pad, BLOCK_SIZE_256>>>(im_d_xgPadded, im_d_cxxPadded, im_d_ygPadded, im_d_cyyPadded, iPaddedSize,
		im_d_cxxPadded, im_d_cyyPadded);
	getLastCudaError("complex_pointwise_multiplication_2d_kernel Launch Failed!");
	// Inverse FFT
	checkCudaErrors(hipfftExecC2C(m_planPadded, im_d_cxxPadded, im_d_cxxPadded, HIPFFT_BACKWARD));
	checkCudaErrors(hipfftExecC2C(m_planPadded, im_d_cyyPadded, im_d_cyyPadded, HIPFFT_BACKWARD));

	// Update the compensated results
	update_final_cxx_cyy_phaseComp_b_kernel<<<blocksImg, threads>>>(
		im_d_cxxPadded, im_d_cyyPadded, d_z.m_d_r, d_z.m_d_phase,
		m_iWidth, m_iHeight, m_iPaddedWidth, m_iPaddedHeight, m_rxxg_norm2, m_ryyg_norm2, m_rSigmaX, m_rSigmaY, m_iSx, m_iSy,
		d_z.m_d_cxx, d_z.m_d_cyy, d_z.m_d_phase_comp, d_z.m_d_b);
	getLastCudaError("update_final_cxx_cyy_phaseComp_b_kernel Launch Failed!");
}

int WFT2_CUDAF::cuWFT2_Initialize(WFT2_DeviceResultsF &d_z)
{
	/*----------------------------WFF&WFR Common parameters initialization-----------------------------*/
	// Half of the Gaussian Window size
	m_iSx = int(round(3 * m_rSigmaX));
	m_iSy = int(round(3 * m_rSigmaY));
	// Guassian Window Size
	m_iWinHeight = 2 * m_iSy + 1;
	m_iWinWidth = 2 * m_iSx + 1;

	// Calculate the initial padding in order to perform the cyclic convolution using FFT
	// The padding size is size(A) + size(B) - 1;
	m_iPaddedHeight = m_iHeight + m_iWinHeight - 1;
	m_iPaddedWidth = m_iWidth + m_iWinWidth - 1;

	// Calculate the second padding in order to fit the optimized size for FFT
	int iH = getFirstGreater(m_iPaddedHeight);
	int iW = getFirstGreater(m_iPaddedWidth);
	if (-1 == iH || -1 == iW)
	{
		// Out of range
		return -1;
	}
	else
	{
		m_iPaddedHeight = OPT_FFT_SIZE[iH];
		m_iPaddedWidth = OPT_FFT_SIZE[iW];

		int iPaddedSize = m_iPaddedHeight * m_iPaddedWidth;

		/* Memory Preallocation on Device */
		// Allocate memory for input padded f which is pre-copmuted and remain unchanged
		checkCudaErrors(hipMalloc((void**)&m_d_fPadded, sizeof(hipfftComplex)*iPaddedSize));
		checkCudaErrors(hipMalloc((void**)&m_d_xf, sizeof(hipfftReal)*iPaddedSize));
		checkCudaErrors(hipMalloc((void**)&m_d_yf, sizeof(hipfftReal)*iPaddedSize));

		/* Make the CUFFT plans */
		checkCudaErrors(hipfftPlan2d(&m_planPadded, m_iPaddedHeight, m_iPaddedWidth, HIPFFT_C2C));
		checkCudaErrors(hipfftSetStream(m_planPadded, 0));

		/* Construct the xf & yf */
		dim3 threads(BLOCK_SIZE_16, BLOCK_SIZE_16);
		dim3 blocks((m_iPaddedWidth + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16, (m_iPaddedHeight + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16);
		// Generate xf, yf
		gen_xf_yf_Kernel<<<blocks, threads>>>(m_d_xf, m_d_yf, m_iPaddedWidth, m_iPaddedHeight);
		getLastCudaError("gen_xf_yf_Kernel Launch Failed!");

		// Shift xf, yf to match the FFT's results
		fftshift_xf_yf_kernel<<<blocks, threads>>>(m_d_xf, m_d_yf, m_iPaddedWidth, m_iPaddedHeight);
		getLastCudaError("fftshift_xf_yf_kernel Launch Failed!");

		/*----------------------------------Specific Inititialization for WFF2&WFR2--------------------------------*/
		if (WFT_FPA::WFT::WFT_TYPE::WFF == m_type)
		{
			cuWFF2_Init(d_z);
		}
		else if (WFT_TYPE::WFR == m_type)
		{
			cuWFR2_Init(d_z);
		}
	}

	return 0;
}
void WFT2_CUDAF::cuWFF2_Init(WFT2_DeviceResultsF &d_z)
{
	int iImageSize = m_iWidth * m_iHeight;
	int iPaddedSize = m_iPaddedHeight * m_iPaddedWidth;

	// Allocate memory for the final results
	checkCudaErrors(hipMalloc((void**)&d_z.m_d_filtered, sizeof(hipfftComplex)*iImageSize));
	
	// 1. Allocate memory for intermediate results per-stream
	// 2. Create CUDA streams 
	// 3. Make the CUFFT plans for each stream
	im_d_Fg = (hipfftComplex**)malloc(m_iNumStreams * sizeof(hipfftComplex*));
	im_d_filtered = (hipfftComplex**)malloc(m_iNumStreams * sizeof(hipfftComplex*));
	im_d_Sf = (hipfftComplex**)malloc(m_iNumStreams * sizeof(hipfftComplex*));
	
	m_cudaStreams = (hipStream_t*)malloc(m_iNumStreams*sizeof(hipStream_t));
	m_planStreams = (hipfftHandle*)malloc(sizeof(hipfftHandle)*m_iNumStreams);

	for (int i = 0; i < m_iNumStreams; i++)
	{
		checkCudaErrors(hipStreamCreate(&(m_cudaStreams[i])));

		checkCudaErrors(hipMalloc((void**)&im_d_Fg[i], sizeof(hipfftComplex)*iPaddedSize));
		checkCudaErrors(hipMalloc((void**)&im_d_filtered[i], sizeof(hipfftComplex)*iPaddedSize));
		checkCudaErrors(hipMalloc((void**)&im_d_Sf[i], sizeof(hipfftComplex)*iPaddedSize));

		checkCudaErrors(hipfftPlan2d(&m_planStreams[i], m_iPaddedHeight, m_iPaddedWidth, HIPFFT_C2C));
		checkCudaErrors(hipfftSetStream(m_planStreams[i], m_cudaStreams[i]));
	}

	if (m_rThr < 0)
	{
		checkCudaErrors(hipMalloc((void**)&m_d_rThr, sizeof(float)));
	}
}
void WFT2_CUDAF::cuWFR2_Init(WFT2_DeviceResultsF &d_z)
{
	int iPaddedSize = m_iPaddedHeight * m_iPaddedWidth;
	int iWinSize = m_iWinHeight * m_iWinWidth;
	int iImgSize = m_iWidth * m_iHeight;

	// Allocate memory for the final results
	checkCudaErrors(hipMalloc((void**)&d_z.m_d_wx, sizeof(hipfftReal)*iImgSize));
	checkCudaErrors(hipMalloc((void**)&d_z.m_d_wy, sizeof(hipfftReal)*iImgSize));
	checkCudaErrors(hipMalloc((void**)&d_z.m_d_r, sizeof(hipfftReal)*iImgSize));
	checkCudaErrors(hipMalloc((void**)&d_z.m_d_phase, sizeof(hipfftReal)*iImgSize));
	checkCudaErrors(hipMalloc((void**)&d_z.m_d_cxx, sizeof(hipfftReal)*iImgSize));
	checkCudaErrors(hipMalloc((void**)&d_z.m_d_cyy, sizeof(hipfftReal)*iImgSize));
	checkCudaErrors(hipMalloc((void**)&d_z.m_d_phase_comp, sizeof(hipfftReal)*iImgSize));
	checkCudaErrors(hipMalloc((void**)&d_z.m_d_b, sizeof(hipfftReal)*iImgSize));

	// 1. Allocate memory for intermediate results per-stream
	// 2. Create CUDA streams 
	// 3. Make the CUFFT plans for each stream
	im_d_wx = (hipfftReal**)malloc(m_iNumStreams*sizeof(hipfftReal*));
	im_d_wy = (hipfftReal**)malloc(m_iNumStreams*sizeof(hipfftReal*));
	im_d_p = (hipfftReal**)malloc(m_iNumStreams*sizeof(hipfftReal*));
	im_d_r = (hipfftReal**)malloc(m_iNumStreams*sizeof(hipfftReal*));
	im_d_Fg = (hipfftComplex**)malloc(m_iNumStreams * sizeof(hipfftComplex*));
	im_d_Sf = (hipfftComplex**)malloc(m_iNumStreams * sizeof(hipfftComplex*));

	m_cudaStreams = (hipStream_t*)malloc(m_iNumStreams*sizeof(hipStream_t));
	m_planStreams = (hipfftHandle*)malloc(sizeof(hipfftHandle)*m_iNumStreams);

	for (int i = 0; i < m_iNumStreams; i++)
	{
		checkCudaErrors(hipStreamCreate(&(m_cudaStreams[i])));

		// Allocate memory for the intermediate arrays
		checkCudaErrors(hipMalloc((void**)&im_d_Fg[i], sizeof(hipfftComplex)*iPaddedSize));
		checkCudaErrors(hipMalloc((void**)&im_d_Sf[i], sizeof(hipfftComplex)*iPaddedSize));
		checkCudaErrors(hipMalloc((void**)&im_d_wx[i], sizeof(hipfftReal)*iImgSize));
		checkCudaErrors(hipMalloc((void**)&im_d_wy[i], sizeof(hipfftReal)*iImgSize));
		checkCudaErrors(hipMalloc((void**)&im_d_p[i], sizeof(hipfftReal)*iImgSize));
		checkCudaErrors(hipMalloc((void**)&im_d_r[i], sizeof(hipfftReal)*iImgSize));		

		checkCudaErrors(hipfftPlan2d(&m_planStreams[i], m_iPaddedHeight, m_iPaddedWidth, HIPFFT_C2C));
		checkCudaErrors(hipfftSetStream(m_planStreams[i], m_cudaStreams[i]));
	}
	// Allocate the memory for corresponding arrays
	checkCudaErrors(hipMalloc((void**)&im_d_g, sizeof(hipfftReal)*iWinSize));
	checkCudaErrors(hipMalloc((void**)&im_d_cyyPadded, sizeof(hipfftComplex)*iPaddedSize));
	checkCudaErrors(hipMalloc((void**)&im_d_cxxPadded, sizeof(hipfftComplex)*iPaddedSize));
	checkCudaErrors(hipMalloc((void**)&im_d_xgPadded, sizeof(hipfftComplex)*iPaddedSize));
	checkCudaErrors(hipMalloc((void**)&im_d_ygPadded, sizeof(hipfftComplex)*iPaddedSize));
	checkCudaErrors(hipMalloc((void**)&m_d_rg_norm2, sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&m_d_rxxg_norm2, sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&m_d_ryyg_norm2, sizeof(float)));

	// Pre-compute g, x.*g, y.*g
	dim3 blocks_g((m_iWinWidth + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16, (m_iWinHeight + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16);
	dim3 threads(BLOCK_SIZE_16, BLOCK_SIZE_16);
	precompute_g_kernel<<<blocks_g, threads>>>(im_d_g, m_iWinWidth, m_iWinHeight, m_rSigmaX, m_rSigmaY);
	getLastCudaError("precompute_g_kernel Launch Failed!");

	int blocks_g1D = std::min((iWinSize + BLOCK_SIZE_256 - 1) / BLOCK_SIZE_256, 2048);
	precompute_norm2g_kernel<<<blocks_g1D, BLOCK_SIZE_256>>>(im_d_g, iWinSize, m_d_rg_norm2);
	getLastCudaError("precompute_norm2g_kernel Launch Failed!");

	precompute_normalized_g_kernel<<<blocks_g1D, BLOCK_SIZE_256>>>(m_d_rg_norm2, iWinSize, im_d_g);
	getLastCudaError("precompute_normalized_g_kernel Launch Failed!");

	dim3 blocks_xyg((m_iPaddedWidth + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16, (m_iPaddedHeight + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16);
	precompute_xg_yg_kernel<<<blocks_xyg, threads>>>(im_d_g, m_iWinWidth, m_iWinHeight, m_iPaddedWidth, m_iPaddedHeight, im_d_xgPadded, im_d_ygPadded);
	getLastCudaError("precompute_xg_yg_kernel Launch Failed!");

	precompute_sum_xxg_yyg_kernel<<<blocks_g1D, BLOCK_SIZE_256>>>(im_d_xgPadded, im_d_ygPadded, m_iWinWidth, m_iWinHeight, m_iPaddedWidth, m_iPaddedHeight, m_d_rxxg_norm2, m_d_ryyg_norm2);
	getLastCudaError("precompute_sum_xxg_yyg_kernel Launch Failed!");

	checkCudaErrors(hipMemcpy(&m_rxxg_norm2, m_d_rxxg_norm2, sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&m_ryyg_norm2, m_d_ryyg_norm2, sizeof(float), hipMemcpyDeviceToHost));

	// Free the im_d_g since it's no need furthermore
	checkCudaErrors(hipFree(im_d_g));	im_d_g = nullptr;

	// Compute the FFT of x.*g & y.*g
	checkCudaErrors(hipfftExecC2C(m_planPadded, im_d_xgPadded, im_d_xgPadded, HIPFFT_FORWARD));
	checkCudaErrors(hipfftExecC2C(m_planPadded, im_d_ygPadded, im_d_ygPadded, HIPFFT_FORWARD));
}

void WFT2_CUDAF::cuWFT2_feed_fPadded(hipfftComplex *d_f)
{
	dim3 threads(BLOCK_SIZE_16, BLOCK_SIZE_16);
	dim3 blocks((m_iPaddedWidth + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16, (m_iPaddedHeight + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16);

	feed_fPadded_kernel<<<blocks, threads>>>(d_f, m_d_fPadded, m_iWidth, m_iHeight, m_iPaddedWidth, m_iPaddedHeight);
	getLastCudaError("feed_fPadded_kernel Launch Failed!");
}
void WFT2_CUDAF::cuWFF2_SetThreashold(hipfftComplex *d_f)
{
	// Set the m_rThr if not set
	if (m_rThr < 0)
	{
		int iImgSize = m_iWidth * m_iHeight;

		// Launch the kernel to compute the threshold
		int blocks = std::min((iImgSize + BLOCK_SIZE_256 - 1) / BLOCK_SIZE_256, 2048);
		compute_WFF_threshold_kernel<<<blocks, BLOCK_SIZE_256>>>(d_f, m_d_rThr, iImgSize);
		getLastCudaError("compute_WFF_threshold_kernel Launch Failed!");

		// Passing back to host
		checkCudaErrors(hipMemcpy(&m_rThr, m_d_rThr, sizeof(float), hipMemcpyDeviceToHost));
		m_rThr = 6 * sqrt(m_rThr *(1.0f / float(iImgSize)) / 3.0f);
	}
}

/*-----------------------------------------/End WFT2 Implementations-------------------------------------------*/

}	// namespace WFT_FPA
}	// namespace WFT