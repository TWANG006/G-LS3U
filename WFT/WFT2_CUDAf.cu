#include "hip/hip_runtime.h"
#include "WFT2_CUDAf.h"
#include ""
#include "mem_manager.h"

#include <iostream>
#define _USE_MATH_DEFINES
#include <math.h>

namespace WFT_FPA{
namespace WFT{

/*---------------------------------------------CUDA Kernels-------------------------------------------------*/
/* 
 PURPOSE: 
	1. Generate the xf and yf for analytically computation of the Gaussian Window in Fourier Domain 
	[yf xf]=meshgrid(-fix(nn/2):nn-fix(nn/2)-1,-fix(mm/2):mm-fix(mm/2)-1); mm, nn are padded height&width
 NOTE: 
	Currently only even size in each dimension is supported
 INPUTS:
	xf, yf: meshgrid in frequency domain
	width, height: width and height of the xf and yf matrices
 OUTPUTS:
	xf, yf: Generated meshgrid
  */
__global__ void gen_xf_yf_Kernel(hipfftReal *xf, hipfftReal *yf, int iWidth, int iHeight)
{
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	int j = threadIdx.x + blockIdx.x * blockDim.x;

	int id = i*iWidth + j;

	int iHalfy = iHeight / 2;
	int iHalfx = iWidth / 2;

	if (i < iHeight && j < iWidth)
	{
		xf[id] = j - iHalfx;
		yf[id] = i - iHalfy;
	}
}
/*
 PURPOSE:
	2. Do the fftshift on xf and yf to be coincide with the CUFFT's results
 NOTE:
	Currently only even size in each dimension is supported 
 INPUTS:
	xf, yf: meshgrid in frequency domian
	width, height: width and height of the xf and yf matrices
 OUTPUTS:
	xf, yf: In-place fft-shifted xf, yf

*/
__global__ void fftshift_xf_yf_kernel(hipfftReal *xf, hipfftReal *yf, int iWidth, int iHeight)
{
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	int j = threadIdx.x + blockIdx.x * blockDim.x;

	int id = i*iWidth + j;

	int iHalfx = iWidth / 2;
	int iHalfy = iHeight / 2;
	int iSlice = iWidth * iHeight;

	int idQ13 = iSlice / 2 + iHalfx;
	int idQ24 = iSlice / 2 - iHalfx;

	hipfftReal Tempx, Tempy;

	if (j < iHalfx)
	{
		if(i < iHalfy)
		{
			Tempx = xf[id];
			Tempy = yf[id];

			// First Quadrant
			xf[id] = xf[id + idQ13];
			yf[id] = yf[id + idQ13];

			// Third Quadrant
			xf[id + idQ13] = Tempx;
			yf[id + idQ13] = Tempy;
		}
	}
	else
	{
		if (i < iHalfy)
		{
			Tempx = xf[id];
			Tempy = yf[id];

			// Second Quadrant
			xf[id] = xf[id + idQ24];
			yf[id] = yf[id + idQ24];

			// Fourth Quadrant
			xf[id + idQ24] = Tempx;
			yf[id + idQ24] = Tempy;
		}
	}
}

/*-------------------------------------------WFT2 Implementations-------------------------------------------*/
WFT2_CUDAF::WFT2_CUDAF(
	int iWidth, int iHeight,
	WFT_TYPE type,
	WFT2_DeviceResultsF& z,
	int iNumStreams)
	: m_iWidth(iWidth)
	, m_iHeight(iHeight)
	, m_type(type)
	, m_rThr(-1)
	, m_d_fPadded(nullptr)
	, m_d_xf(nullptr)
	, m_d_yf(nullptr)
{
	// Check the input image size
	if (iWidth % 2 != 0 || iHeight % 2 != 0)
	{
		std::cout << "GPU implementation of WFT curretly only suppports even image size!" << std::endl;
		throw -2;

	}
	
	// Get the number of SMs on GPU 
	hipDeviceGetAttribute(&m_iSMs, hipDeviceAttributeMultiprocessorCount, 0);

	/* Type specific parameter initializations*/
	if (WFT_TYPE::WFF == m_type)
	{
		m_rSigmaX = 10.0f;
		m_rWxl = -2.0f - 3.0f / m_rSigmaX;
		m_rWxi = 1.0f / m_rSigmaX;
		m_rWxh = 2.0f + 3.0f / m_rSigmaX;

		m_rSigmaY = 10.0f;
		m_rWyl = -2.0f - 3.0f / m_rSigmaY;
		m_rWyi = 1.0f / m_rSigmaY;
		m_rWyh = 2.0f + 3.0f / m_rSigmaY;
	}
	else if (WFT_TYPE::WFR == m_type)
	{
		m_rSigmaX = 10.0f;
		m_rWxl = -2.0f;
		m_rWxi = 0.025f;
		m_rWxh = 2.0f;

		m_rSigmaY = 10.0f;
		m_rWyl = -2.0f;
		m_rWyi = 0.025f;
		m_rWyh = 2.0f;
	}

	// scale for window so that norm2 of the window is 1. 
	m_rGaussianNorm2 = sqrt(4 * float(M_PI)*m_rSigmaX*m_rSigmaY);

	/* Do the Initialization */
	if(-1 == cuWFT2_Initialize(z))
	{
		std::cout<<"FFT padding is out of range [4096]. Shrink the size of either the image or the Gaussian Window!"<<std::endl;
		throw -1;
	}
}

WFT2_CUDAF::WFT2_CUDAF(
	int iWidth, int iHeight,
	WFT_TYPE type,
	float rSigmaX, float rWxl, float rWxh, float rWxi,
	float rSigmaY, float rWyl, float rWyh, float rWyi,
	float rThr,
	WFT2_DeviceResultsF &z,
	int iNumStreams)
	: m_iWidth(iWidth)
	, m_iHeight(iHeight)
	, m_type(type)
	, m_rSigmaX(rSigmaX)
	, m_rSigmaY(rSigmaY)
	, m_rWxl(rWxl)
	, m_rWxi(rWxi)
	, m_rWxh(rWxh)
	, m_rWyl(rWxl)
	, m_rWyi(rWyi)
	, m_rWyh(rWyh)
	, m_rThr(rThr)
	, m_d_fPadded(nullptr)
	, m_d_xf(nullptr)
	, m_d_yf(nullptr)
{
	// Check the input image size
	if (iWidth % 2 != 0 || iHeight % 2 != 0)
	{
		std::cout << "GPU implementation of WFT curretly only suppports even image size!" << std::endl;
		throw -2;

	}
	
	// Get the number of SMs on GPU 
	hipDeviceGetAttribute(&m_iSMs, hipDeviceAttributeMultiprocessorCount, 0);

	// scale for window so that norm2 of the window is 1. 
	m_rGaussianNorm2 = sqrt(4 * float(M_PI)*m_rSigmaX*m_rSigmaY);
	/* Do the Initialization */
	if (-1 == cuWFT2_Initialize(z))
	{
		std::cout << "FFT padding is out of range [4096]. Shrink the size of either the image or the Gaussian Window!" << std::endl;
		throw - 1;
	}
}

WFT2_CUDAF::~WFT2_CUDAF()
{
	WFT_FPA::Utils::cudaSafeFree(m_d_fPadded);
	WFT_FPA::Utils::cudaSafeFree(m_d_xf);
	WFT_FPA::Utils::cudaSafeFree(m_d_yf);

	hipfftDestroy(m_planForwardf);
}

void WFT2_CUDAF::operator()(
	hipfftComplex *f,
	WFT2_DeviceResultsF &z,
	double &time)
{
	
}


/* Private functions */

void WFT2_CUDAF:: cuWFF2(hipfftComplex *f, WFT2_DeviceResultsF &z, double &time)
{
}
void WFT2_CUDAF::cuWFR2(hipfftComplex *f, WFT2_DeviceResultsF &z, double &time)
{
}

int WFT2_CUDAF::cuWFT2_Initialize(WFT2_DeviceResultsF &z)
{
	/*----------------------------WFF&WFR Common parameters initialization-----------------------------*/
	// Half of the Gaussian Window size
	m_iSx = int(round(3 * m_rSigmaX));
	m_iSy = int(round(3 * m_rSigmaY));
	// Guassian Window Size
	m_iWinHeight = 2 * m_iSy + 1;
	m_iWinWidth = 2 * m_iSx + 1;

	// Calculate the initial padding in order to perform the cyclic convolution using FFT
	// The padding size is size(A) + size(B) - 1;
	m_iPaddedHeight = m_iHeight + m_iWinHeight - 1;
	m_iPaddedWidth = m_iWidth + m_iWinWidth - 1;

	// Calculate the second padding in order to fit the optimized size for FFT
	int iH = getFirstGreater(m_iPaddedHeight);
	int iW = getFirstGreater(m_iPaddedWidth);
	if (-1 == iH || -1 == iW)
	{
		// Out of range
		return -1;
	}
	else
	{
		m_iPaddedHeight = OPT_FFT_SIZE[iH];
		m_iPaddedWidth = OPT_FFT_SIZE[iW];

		int iPaddedSize = m_iPaddedHeight * m_iPaddedWidth;

		/* Memory Preallocation on Device */
		// Allocate memory for input padded f which is pre-copmuted and remain unchanged
		checkCudaErrors(hipMalloc((void**)&m_d_fPadded, sizeof(hipfftComplex)*iPaddedSize));
		checkCudaErrors(hipMalloc((void**)&m_d_xf, sizeof(hipfftReal)*iPaddedSize));
		checkCudaErrors(hipMalloc((void**)&m_d_yf, sizeof(hipfftReal)*iPaddedSize));

		/* Make the CUFFT plan for the precomputation of Ff = fft2(f) */
		checkCudaErrors(hipfftPlan2d(&m_planForwardf, m_iPaddedWidth, m_iPaddedHeight, HIPFFT_C2C));
		
		/* Construct the xf & yf */
		dim3 threads(BLOCK_SIZE_16, BLOCK_SIZE_16);
		dim3 blocks((m_iPaddedWidth + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16, (m_iPaddedHeight + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16);
		// Generate xf, yf
		gen_xf_yf_Kernel<<<blocks, threads>>>(m_d_xf, m_d_yf, m_iPaddedWidth, m_iPaddedHeight);
		getLastCudaError("gen_xf_yf_Kernel Launch Failed!");
		// Shift xf, yf to match the FFT's results
		fftshift_xf_yf_kernel<<<blocks, threads>>>(m_d_xf, m_d_yf, m_iPaddedWidth, m_iPaddedHeight);
		getLastCudaError("fftshift_xf_yf_kernel Launch Failed!");
	}

	return 0;
}

void WFT2_CUDAF::cuWFF2_Init(WFT2_DeviceResultsF &z)
{
}

int WFT2_CUDAF:: cuWFR2_Init(WFT2_DeviceResultsF &z)
{
	return 0;
}

void WFT2_CUDAF::cuWFT2_feed_fPadded(hipfftComplex *f)
{
}
void WFT2_CUDAF::cuWFF2_SetThreashold(hipfftComplex *f)
{
}


}	// namespace WFT_FPA
}	// namespace WFT